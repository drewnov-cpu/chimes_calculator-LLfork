#include "hip/hip_runtime.h"
/* 
    ChIMES Calculator
    Copyright (C) 2020 Rebecca K. Lindsey, Nir Goldman, and Laurence E. Fried
    Contributing Author:  Rebecca K. Lindsey (2020) 
*/

#include<vector>
#include<iostream>
#include<iomanip>
#include<fstream>
#include<string>
#include<sstream>
#include<cstdlib>
#include<algorithm>
#include<cmath>
#include<map>

using namespace std;

#include "chimesFF.h"
#include "gpu_compute.h"   

template <typename T>
int get_index(const vector<T>  & vec, const T  & element)
{
    auto it = find(vec.begin(), vec.end(), element);
 
    if (it != vec.end())
    {
        return distance(vec.begin(), it);
    }
    else
    {
        cout << "chimesFF: " << "ERROR: Could not find element in vector" << endl;
        exit(0);
    }
}

template <typename T>
int get_index_if(const vector<T>  & vec, const T  & element, vector<bool> & disqualified)
{

    if (disqualified.size() != vec.size())
    {
        cout << "chimesFF: " << "ERROR: get_index_if(...): Qualification criteria does not match vector length" << endl;
        cout << "chimesFF: " << "vec.size(): " << vec.size() << endl;
        cout << "chimesFF: " << "disqualified.size(): " << disqualified.size() << endl;
        exit(0);
    }

    for(int i=0; i<vec.size(); i++)
    {
        if ((vec[i]==element) && (!disqualified[i]))
        {
            disqualified[i] = true;
            return i;
        }
    }

    cout << "chimesFF: " << "ERROR: Could not find element in vector: " << element << endl;
    
    for(int i=0; i<vec.size(); i++)
        cout << "chimesFF: " << "\t" << vec[i] << " " << disqualified[i] << endl;
    
    exit(0);
}

int chimesFF::get_proper_pair(string ty1, string ty2)
{

    for(int i=0; i<pair_params_atm_chem_1.size(); i++)
    {
        if (ty1 == pair_params_atm_chem_1[i])
            if (ty2 == pair_params_atm_chem_2[i])
                return i;
        
        if (ty2 == pair_params_atm_chem_1[i])
            if (ty1 == pair_params_atm_chem_2[i])
                return i;
    }
            
    cout << "chimesFF: " << "ERROR: No proper pair name found for atom types" << ty1 << ", " << ty2 << endl;
    exit(0);
}

chimesFF::chimesFF()
{
    natmtyps = 0;
    penalty_params.resize(2);
    
    // Set defaults
    
    fcut_type = fcutType::CUBIC ;
    
    penalty_params[0] = 0.01;
    penalty_params[1] = 1.0E4;

    inner_smooth_distance = 0.01 ;
	
}
chimesFF::~chimesFF(){}

void chimesFF::init(int mpi_rank)
{
    rank = mpi_rank;
    print_pretty_stuff();
}

void chimesFF::print_pretty_stuff()
{
    if (rank == 0)
    {
        cout << "chimesFF: " <<  endl;
        cout << "chimesFF: " << "01000011011010001001001010011010100010101010011 0100010101101110110011101101001011011101100101  " << endl;
        cout << "chimesFF: " <<  endl;
        cout << "chimesFF: " << "      _____  _      _____  __  __  ______   _____   ______                _                      " << endl;
        cout << "chimesFF: " << "     / ____|| |    |_   _||  \\/  ||  ____| / ____| |  ____|              (_)                    " << endl;
        cout << "chimesFF: " << "    | |     | |__    | |  | \\  / || |__   | (___   | |__    _ __    __ _  _  _ __    ___        " << endl;
        cout << "chimesFF: " << "    | |     | '_ \\   | |  | |\\/| ||  __|   \\___ \\  |  __|  | '_ \\  / _` || || '_ \\  / _ \\ " << endl;
        cout << "chimesFF: " << "    | |____ | | | | _| |_ | |  | || |____  ____) | | |____ | | | || (_| || || | | ||  __/        " << endl;
        cout << "chimesFF: " << "     \\_____||_| |_||_____||_|  |_||______||_____/  |______||_| |_| \\__, ||_||_| |_| \\___|     " << endl;
        cout << "chimesFF: " << "                                                                    __/ |                        " << endl;
        cout << "chimesFF: " << "                                                                   |___/                         " << endl;  
        cout << "chimesFF: " << endl;
        cout << "chimesFF: " << "                     Copyright (C) 2020 R.K. Lindsey, L.E. Fried, N. Goldman                     " << endl;    
        cout << "chimesFF: " << endl;
        cout << "chimesFF: " << "01000011011010001001001010011010100010101010011 0100010101101110110011101101001011011101100101   " << endl;
        cout << "chimesFF: " << endl;
    }
      
}

int chimesFF::split_line(string line, vector<string> & items)
{
    // Break a line up into tokens based on space separators.
    // Returns the number of tokens parsed.
    
    string       contents;
    stringstream sstream;

    // Strip comments beginining with ! or ## and terminal new line

    int pos = line.find('!');
      
    if ( pos != string::npos ) 
        line.erase(pos, line.length() - pos);

    pos = line.find("##");
    if ( pos != string::npos ) 
        line.erase(pos, line.length()-pos);

    pos = line.find('\n');
    if ( pos != string::npos ) 
        line.erase(pos, 1);

    sstream.str(line);
     
    items.clear();

    while ( sstream >> contents ) 
        items.push_back(contents);

    return items.size();
}

string chimesFF::get_next_line(istream& str)
{
    // Read a line and return it, with error checking.
    
    string line;

    getline(str, line);
    
    if ( ! str.good() )
    {
        if (rank == 0)
            cout << "chimesFF: " << "Error reading line" << line << endl;
        exit(0);
    } 

    return line;
}

void chimesFF::read_parameters(string paramfile)
{
    // Open the parameter file, run sanity checks
    
    ifstream param_file;
    param_file.open(paramfile.data());
    
    if (rank == 0)
        cout << "chimesFF: " << "Reading parameters from file: " << paramfile << endl;
    
    if(!param_file.is_open())
    {
        if (rank == 0)
            cout << "chimesFF: " << "ERROR: Cannot open parameter file: " << paramfile << endl;
        exit(0);
    }
    
    // Declare parsing variables
    
    
    bool           found_end = false;
    string         line;
    string         tmp_str;
    vector<string> tmp_str_items;
    int            tmp_no_items;
    int            tmp_int;
    int            no_pairs;
    
    // Check that this is actually a chebyshev parameter set

    while (!found_end)
    {
        line = get_next_line(param_file);

           // Break out of loop

           if(line.find("ENDFILE") != string::npos)
        {
            if (rank == 0)
            {
                cout << "chimesFF: " << "ERROR: Could not find line containing: \" PAIRTYP: CHEBYSHEV\" " << endl;
                cout << "chimesFF: " << "       ...Is this a ChIMES force field parameter file?" << endl;
            }
            exit(0);
        }
        
        if(line.find("PAIRTYP: CHEBYSHEV") != string::npos)
        {
            tmp_no_items = split_line(line, tmp_str_items);
            
            if (tmp_no_items < 3)
            {    
                if (rank == 0)
                    cout << "chimesFF: " << "ERROR: \"PAIRTYP: CHEBYSHEV\" line must at least contain the 2-body order" << endl;
                exit(0);
            }
            
            poly_orders.push_back(stoi(tmp_str_items[2]));
            
            if (tmp_no_items >= 4)
                poly_orders.push_back(stoi(tmp_str_items[3]));

            if (tmp_no_items >= 5)
                poly_orders.push_back(stoi(tmp_str_items[4]));    
            
            while (poly_orders.size() < 3)
                poly_orders.push_back(0);
            
            if (rank == 0)
            {
                cout << "chimesFF: " << "Using respective 2, 3, and 4-body orders of: " << poly_orders[0] << " " << poly_orders[1] << " " << poly_orders[2] << endl;
            
                cout << "chimesFF: " << "Note: Ignoring polynomial domain; assuming [-1,1]" << endl;    
            }
            
            break;    
        }
    }
    
    // If we've made it to here, then this should contain Chebyshev params. Rewind and start looking for general information
        
    param_file.seekg(0);
    
    found_end = false;
    
    while (!found_end)
    {
        line = get_next_line(param_file);
        
           if(line.find("ENDFILE") != string::npos)
            break;        
    
        if(line.find("ATOM TYPES:") != string::npos)
        {
            tmp_no_items = split_line(line, tmp_str_items);
        
            natmtyps = stoi(tmp_str_items[2]);
        
            if (rank == 0)
                cout << "chimesFF: " << "Will consider " << natmtyps << " atom types:" << endl;
                
            energy_offsets.resize(natmtyps);
            
            for(int i=0; i<natmtyps; i++)
                energy_offsets[i] = 0.0;
        }
        
        if(line.find("# TYPEIDX #") != string::npos)
        {
            atmtyps.resize(natmtyps);
			masses.resize(natmtyps);
            for (int i=0; i<natmtyps; i++)
            {
                line = get_next_line(param_file);
                split_line(line, tmp_str_items);
                atmtyps[i] = tmp_str_items[1];
				masses[i]  = stod(tmp_str_items[3]);
                
                if (rank == 0)
                    cout << "chimesFF: " << "\t" << i << " " << atmtyps[i] << endl;
            }
            
        }
            
        if(line.find("ATOM PAIRS:") != string::npos)
        {
            tmp_no_items = split_line(line, tmp_str_items);
        
            no_pairs = stoi(tmp_str_items[2]);
        
            if (rank == 0)
                cout << "chimesFF: " << "Will consider " << no_pairs << " atom pair types" << endl;        
        }    
        
        if(line.find("# PAIRIDX #") != string::npos)
        {
            if(line.find("# USEOVRP #") != string::npos)
                continue;
        
            pair_params_atm_chem_1.resize(no_pairs);
            pair_params_atm_chem_2.resize(no_pairs);
            chimes_2b_cutoff      .resize(no_pairs);
            morse_var             .resize(no_pairs);
            
            ncoeffs_2b            .resize(no_pairs);
            chimes_2b_pows        .resize(no_pairs);
            chimes_2b_params      .resize(no_pairs);
            chimes_2b_cutoff      .resize(no_pairs);

            string tmp_xform_style;
            
            for (int i=0; i<no_pairs; i++)
            {
                line = get_next_line(param_file);
                
                tmp_no_items = split_line(line, tmp_str_items);

                int pair_input_version = 0;
				
                if ( tmp_no_items == 8 )
                {
					if ( rank == 0 && i == 0 ) cout << "chimesFF: Detected version 1 pair specification (with S_DELTA)\n";
					pair_input_version = 1;
                }
                else if ( tmp_no_items == 7 )
                {
					if ( rank == 0 && i == 0 ) cout << "chimesFF: Detected version 2 pair specification (no S_DELTA)\n";
					pair_input_version = 2;
                }
                else
                {
					if ( rank == 0 )
					{
						cout << "Incorrect input in line: " << line << endl;
						cout << "Expect 7 or 8 entries\n";
					}
					exit(0);
                }
            
                pair_params_atm_chem_1[i] = tmp_str_items[1];
                pair_params_atm_chem_2[i] = tmp_str_items[2];
                
                if (rank == 0)
                    cout << "chimesFF: " << "\t" << i << " " << pair_params_atm_chem_1[i] << " " << pair_params_atm_chem_2[i]<< endl;
                
                chimes_2b_cutoff[i].push_back(stod(tmp_str_items[3])); // Inner cutoff    
                chimes_2b_cutoff[i].push_back(stod(tmp_str_items[4])); // Outer cutoff

                int xform_style_idx, morse_idx;
				
                if ( pair_input_version == 1 )
                {
					xform_style_idx = 6;
					morse_idx = 7;
                }
                else if ( pair_input_version == 2 )
                {
					xform_style_idx = 5;
					morse_idx = 6;
                } 
                else
                {
					if ( rank == 0 ) cout << "Bad pair input version\n";
					exit(0);
                }
                    
                if (i==0)
                {
                    tmp_xform_style = tmp_str_items[xform_style_idx];
                }
                else if ( tmp_str_items[xform_style_idx] != tmp_xform_style)    
                {
					if (rank == 0)
						cout << "chimesFF: " << "Distance transformation style must be the same for all pair types" << endl;
					exit(0);
                }

                if (tmp_xform_style == "MORSE" )
                {
					if ( tmp_no_items > morse_idx )
						morse_var[i] = stod(tmp_str_items[morse_idx]);
					else {
						if ( rank == 0 )
							cout << "chimesFF: Missing morse lambda value in line: \n" << line << endl;
						exit(0);
					}
				}
            }
                
            xform_style = tmp_xform_style;
            
            if (rank == 0)
                cout << "chimesFF: " << "Read the following pair type information:" << endl;
            
            for (int i=0; i<no_pairs; i++)
            {
				if (rank == 0)
					cout << "chimesFF: " << "\t" << pair_params_atm_chem_1[i] << " " << pair_params_atm_chem_2[i] << " r_cut_in: " << fixed << right << setprecision(5) << chimes_2b_cutoff[i][0] << " r_cut_out: " << chimes_2b_cutoff[i][1] << " " <<  xform_style;
                
				if (xform_style == "MORSE")
				{
					if (rank == 0)
						cout << " " << morse_var[i] << endl;
				}
				else
					if (rank == 0)
						cout << endl;
			}
        }
            
        if(line.find("FCUT TYPE:") != string::npos)
        {
            tmp_no_items = split_line(line, tmp_str_items);
        
            if ( tmp_str_items[2] == "CUBIC" )
                fcut_type = fcutType::CUBIC ;
            else if ( tmp_str_items[2] == "TERSOFF" )
                fcut_type = fcutType::TERSOFF ;
            else
            {
                if ( rank == 0 ) 
                    cout << "Error: unknown FCUT TYPE: " << tmp_str_items[2] << endl ;
                exit(1) ;
            }
                    
            if (rank == 0)
                cout << "chimesFF: " << "Will use cutoff style " << tmp_str_items[2] << endl ;
            
            if (fcut_type == fcutType::TERSOFF )
            {
                fcut_var = stod(tmp_str_items[3]);
                
                if (rank == 0)
                    cout << " " << fcut_var << endl;
            }
            else
                if (rank == 0)
                    cout << endl;
        }
        
        if(line.find("PAIR CHEBYSHEV PENALTY DIST:") != string::npos)
        {    
            tmp_no_items = split_line(line, tmp_str_items);
            
            penalty_params[0] = stod(tmp_str_items[4]);
            
            if (rank == 0)
                cout << "chimesFF: " << "Will use penalty distance: " << penalty_params[0] << endl;
        }
        
        if(line.find("PAIR CHEBYSHEV PENALTY SCALING:") != string::npos)
        {    
            tmp_no_items = split_line(line, tmp_str_items);
            
            penalty_params[1] = stod(tmp_str_items[4]);
            
            if (rank == 0)
                cout << "chimesFF: " << "Will use penalty scaling: " << penalty_params[1] << endl;
        }
        
        if(line.find("NO ENERGY OFFSETS:") != string::npos)
        {
            int tmp_no = split_line(line, tmp_str_items);
                        
            if(stoi(tmp_str_items[tmp_no-1]) != natmtyps)
            {
                cout << "chimesFF: " << "ERROR: Number of energy offsets do not match number of atom types" << endl;
                exit(0);
            }

            // Expects atom offsets in the same order as atom types were provided originally
            
            if (rank == 0)
                cout << "chimesFF: " << "Will use single atom energy offsets: "<< endl;
            
            int tmp_idx;
            
            for (int i=0; i<natmtyps; i++)
            {
                line = get_next_line(param_file);
                split_line(line, tmp_str_items);
                tmp_idx = stoi(tmp_str_items[2]);
                
                energy_offsets[tmp_idx-1] = stod(tmp_str_items[3]);
                
                if (rank == 0)
                    cout << "chimesFF: " << "\t" << tmp_idx << " " << atmtyps[tmp_idx-1] << " " << energy_offsets[tmp_idx-1] << endl;
            }
            
        }                
    }
    
    // Rewind and read the 2-body Chebyshev pair parameters
    
    param_file.seekg(0);
    
    found_end = false;
    
    while (!found_end)
    {
		line = get_next_line(param_file);

		if(line.find("ENDFILE") != string::npos)
			break;            
        
        if(line.find("PAIRTYPE PARAMS:") != string::npos)
        {
            tmp_no_items = split_line(line, tmp_str_items);
            
            tmp_int = stoi(tmp_str_items[2]);
            
            if (rank == 0)
                cout << "chimesFF: " << "Read 2B parameters for pair: " << tmp_int << " " << tmp_str_items[3] << " " << tmp_str_items[4] << endl;
            
            line = get_next_line(param_file);
            
            split_line(line, tmp_str_items); // Empty line
            
            ncoeffs_2b[tmp_int] = poly_orders[0];
            
            for(int i=0; i<poly_orders[0]; i++)
            {
                line = get_next_line(param_file);
                split_line(line, tmp_str_items);
                
                chimes_2b_pows  [tmp_int].push_back(stoi(tmp_str_items[0]));                
                chimes_2b_params[tmp_int].push_back(stod(tmp_str_items[1]));
                
                if (rank == 0)
                    cout << "chimesFF: " << "\t" << chimes_2b_pows[tmp_int][i] << " " << chimes_2b_params[tmp_int][i] << endl;
            }
        }
        
        if(line.find("PAIRMAPS:") != string::npos)
        {
            // Read the slow map and build the fast map
            
            tmp_no_items = split_line(line, tmp_str_items);
            
            n_pair_maps = stoi(tmp_str_items[1]);
            
            atom_typ_pair_map.resize(n_pair_maps);
            atom_idx_pair_map.resize(n_pair_maps);
            
            atom_int_prpr_map.resize(n_pair_maps);
            
            if (rank == 0)
                cout << "chimesFF: " << "Built the following 2-body pair \"slow\" map:" << endl;
            
            for(int i=0; i<n_pair_maps; i++)
            {
                line = get_next_line(param_file);
                split_line(line, tmp_str_items);
                
                atom_idx_pair_map[i] = stoi(tmp_str_items[0]);
                atom_typ_pair_map[i] =      tmp_str_items[1];
                
                if (rank == 0)
                    cout << "chimesFF: " << "\t" << atom_idx_pair_map[i] << " " << atom_typ_pair_map[i] << "(i: " << i << ")" << endl;

            }

            if (rank == 0)
                cout << "chimesFF: " << "Built the following 2-body pair \"fast\" map:" << endl;
            
            atom_int_pair_map.resize((natmtyps-1)*natmtyps + (natmtyps-1) + 1); // Maximum possible pair value + a small buffer
            

            for(int i=0; i<natmtyps; i++)
            {
                for (int j=0; j<natmtyps; j++)
                {
                    // Get the pair type name for the set of atoms
                    
                    tmp_str = atmtyps[i] + atmtyps[j];

                    tmp_int = get_index(atom_typ_pair_map, tmp_str);
                    
                    atom_int_pair_map[ i*natmtyps + j ] = atom_idx_pair_map[tmp_int];
                    

                    tmp_int = get_proper_pair(atmtyps[i],atmtyps[j]);
                    
                    atom_int_prpr_map [ i*natmtyps + j ] = pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];

                    
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << tmp_str << ": " << i*natmtyps + j << " " << atom_int_pair_map[ i*natmtyps + j ] << endl;

                }
            }                        
        }
    }
    
    // Rewind and read the 3-body Chebyshev pair parameters
    
    if (poly_orders[1] > 0)
    {
        int ntrips;
        int tmp_idx;
        
        // Read parameters
        
        param_file.seekg(0);
        
        found_end = false;
    
        while (!found_end)
        {
            line = get_next_line(param_file);
        
               if(line.find("ENDFILE") != string::npos)
                break;    
            
            if(line.find("ATOM PAIR TRIPLETS:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                ntrips = stoi(tmp_str_items[3]);
                                
                ncoeffs_3b      .resize(ntrips);
                chimes_3b_powers.resize(ntrips);                
                chimes_3b_params.resize(ntrips);
                chimes_3b_cutoff.resize(ntrips);
    
                
                trip_params_atm_chems.resize(ntrips);                
                trip_params_pair_typs.resize(ntrips);
            }
            
            if(line.find("TRIPLETTYPE PARAMS:") != string::npos)
            {
                vector<int> tmp_int_vec(3);

                line = get_next_line(param_file);
                
                split_line(line, tmp_str_items);

                tmp_int = stoi(tmp_str_items[1]);
                
                trip_params_atm_chems[tmp_int].push_back(tmp_str_items[3]);
                trip_params_atm_chems[tmp_int].push_back(tmp_str_items[4]);
                trip_params_atm_chems[tmp_int].push_back(tmp_str_items[5]);

                if (rank == 0)
                    cout << "chimesFF: " << "Read 3B parameters for triplet: " << tmp_int << " " << trip_params_atm_chems[tmp_int][0] << " " << trip_params_atm_chems[tmp_int][1] << " " << trip_params_atm_chems[tmp_int][2] << endl;
                
                line = get_next_line(param_file);
                
                split_line(line, tmp_str_items);
            
                trip_params_pair_typs[tmp_int].push_back(tmp_str_items[1]);
                trip_params_pair_typs[tmp_int].push_back(tmp_str_items[2]);
                trip_params_pair_typs[tmp_int].push_back(tmp_str_items[3]);
		
		// Check for excluded triplet types
	
		if(tmp_str_items[4] != "EXCLUDED:")
		{
                	ncoeffs_3b[tmp_int] = stoi(tmp_str_items[7]);    
	
        	        get_next_line(param_file);
        	        get_next_line(param_file);
            
        	        for(int i=0; i<ncoeffs_3b[tmp_int]; i++)
        	        {
        	            line = get_next_line(param_file);
        	            split_line(line, tmp_str_items);
                    
        	            tmp_int_vec[0] = stoi(tmp_str_items[1]);
        	            tmp_int_vec[1] = stoi(tmp_str_items[2]);
        	            tmp_int_vec[2] = stoi(tmp_str_items[3]);
                    
        	            chimes_3b_powers[tmp_int].push_back(tmp_int_vec);                    
        	            chimes_3b_params[tmp_int].push_back(stod(tmp_str_items[6]));
                
        	            if (rank == 0)
        	                cout << "chimesFF: " << "\t" << chimes_3b_powers[tmp_int][i][0] << " " << chimes_3b_powers[tmp_int][i][1] << " " << chimes_3b_powers[tmp_int][i][2] << " " << chimes_3b_params[tmp_int][i] << endl;
        	        }
		}
		else
		{
			cout << "chimesFF: \tType is excluded... skipping." << endl;
		}
            }    
            
            if(line.find("TRIPMAPS:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                n_trip_maps = stoi(tmp_str_items[1]);
                
                atom_idx_trip_map.resize(n_trip_maps);
                atom_typ_trip_map.resize(n_trip_maps);
                
                if (rank == 0)                
                    cout << "chimesFF: " << "Built the following 3-body pair \"slow\" map:" << endl;
            
                for(int i=0; i<n_trip_maps; i++)
                {
                    line = get_next_line(param_file);
                    split_line(line, tmp_str_items);
                
                    atom_idx_trip_map[i] = stoi(tmp_str_items[0]);
                    atom_typ_trip_map[i] =      tmp_str_items[1];
                
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << atom_idx_trip_map[i] << " " << atom_typ_trip_map[i] << endl;
                }        
                
                if (rank == 0)
                    cout << "chimesFF: " << "Built the following 3-body pair \"fast\" map:" << endl;

                atom_int_trip_map.resize(natmtyps*natmtyps*natmtyps);
                
                for(int i=0; i<natmtyps; i++)
                {
                    for (int j=0; j<natmtyps; j++)
                    {
                        for(int k=0; k<natmtyps; k++)
                        {
                            // Get the trip type name for the set of atoms
                            
                            tmp_str = "";

                            tmp_int  = get_proper_pair(atmtyps[i], atmtyps[j]);
                            tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];    
                
                            tmp_int  = get_proper_pair(atmtyps[i], atmtyps[k]);
                            tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];    
                
                            tmp_int  = get_proper_pair(atmtyps[j], atmtyps[k]);
                            tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];            
                                            
                            tmp_int = get_index(atom_typ_trip_map, tmp_str);

                            tmp_idx = i*natmtyps*natmtyps + j*natmtyps + k;

                            atom_int_trip_map[ tmp_idx ] = atom_idx_trip_map[tmp_int];
                                                        
                            if (rank == 0)
                                cout << "chimesFF: " << "\t" << tmp_idx << " " << atom_int_trip_map[ tmp_idx  ]  << endl;
                        }
                    }
                }
            }            
        }
        
        // Set up cutoffs ... First set to match 2-body, then read special if they exist
        
        int atmtyp_1,  atmtyp_2,  atmtyp_3;
        int pairtyp_1, pairtyp_2, pairtyp_3;

        for(int i=0; i<ntrips; i++) 
        {
            // Figure out the atom type index for each atom in the triplet type 
                        
            atmtyp_1 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), trip_params_atm_chems[i][0]));    
            atmtyp_2 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), trip_params_atm_chems[i][1]));    
            atmtyp_3 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), trip_params_atm_chems[i][2]));    
                        
            // Figure out the corresponding 2-body pair type
            
            pairtyp_1 = atom_int_pair_map[ atmtyp_1*natmtyps + atmtyp_2 ];
            pairtyp_2 = atom_int_pair_map[ atmtyp_1*natmtyps + atmtyp_3 ];
            pairtyp_3 = atom_int_pair_map[ atmtyp_2*natmtyps + atmtyp_3 ];
    
            // Set the default inner/outer cutoffs to the corresponding 2-body value

            chimes_3b_cutoff[i].resize(2);

            chimes_3b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_1][0]);
            chimes_3b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_2][0]);
            chimes_3b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_3][0]);
            
            chimes_3b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_1][1]);
            chimes_3b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_2][1]);
            chimes_3b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_3][1]);                    
        }
        
        param_file.seekg(0);
        
        int    nentries;
        double cutval;
        
        found_end = false;
        
        while (!found_end)
        {
            line = get_next_line(param_file);
        
			if(line.find("ENDFILE") != string::npos)
                break;                
            
            if(line.find("SPECIAL 3B S_MAXIM:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                if (rank == 0)
                    cout << "chimesFF: " << "Set the following special 3-body outer cutoffs: " << endl;
                
                if(tmp_str_items[3] == "ALL")
                {
                    cutval = stod(tmp_str_items[4]);
                                        
                    for(int i=0; i<ntrips; i++)
                    {
                        chimes_3b_cutoff[i][1][0] = cutval;
                        chimes_3b_cutoff[i][1][1] = cutval;
                        chimes_3b_cutoff[i][1][2] = cutval;                    
                    }
                }
                else
                {
                    nentries = stoi(tmp_str_items[4]);
                    
                    vector<string> pair_name(3);
                    vector<double> cutoffval(3);

    
                    for(int i=0; i<nentries; i++)
                    {
                        line = get_next_line(param_file);
                        
                        split_line(line, tmp_str_items);
                        
                        tmp_int = atom_idx_trip_map[distance(atom_typ_trip_map.begin(), find(atom_typ_trip_map.begin(), atom_typ_trip_map.end(), tmp_str_items[0]))];

                        pair_name[0] = tmp_str_items[1];
                        pair_name[1] = tmp_str_items[2];
                        pair_name[2] = tmp_str_items[3];
                        
                        cutoffval[0] = stod(tmp_str_items[4]);
                        cutoffval[1] = stod(tmp_str_items[5]);
                        cutoffval[2] = stod(tmp_str_items[6]);
                        
                        vector<bool>   disqualified(3,false);
                        
                        chimes_3b_cutoff[tmp_int][1][ get_index_if(trip_params_pair_typs[tmp_int], pair_name[0], disqualified) ] = cutoffval[0];
                        chimes_3b_cutoff[tmp_int][1][ get_index_if(trip_params_pair_typs[tmp_int], pair_name[1], disqualified) ] = cutoffval[1];
                        chimes_3b_cutoff[tmp_int][1][ get_index_if(trip_params_pair_typs[tmp_int], pair_name[2], disqualified) ] = cutoffval[2];
                                        
                    }
                }
                
                for(int i=0; i<ntrips; i++)
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << i << " " << chimes_3b_cutoff[i][1][0] << " " << chimes_3b_cutoff[i][1][1] << " " << chimes_3b_cutoff[i][1][2] << endl;
                
            }

            if(line.find("SPECIAL 3B S_MINIM:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                if (rank == 0)
                    cout << "chimesFF: " << "Set the following special 3-body inner cutoffs: " << endl;
                
                if(tmp_str_items[3] == "ALL")
                {
                    cutval = stod(tmp_str_items[4]);
                    
                    for(int i=0; i<ntrips; i++)
                    {
                        chimes_3b_cutoff[i][0][0] = cutval;
                        chimes_3b_cutoff[i][0][1] = cutval;
                        chimes_3b_cutoff[i][0][2] = cutval;
                        
                    }
                }
                else
                {
                    nentries = stoi(tmp_str_items[4]);
                    
                    vector<string> pair_name(3);
                    vector<double> cutoffval(3);


                    for(int i=0; i<nentries; i++)
                    {
                        line = get_next_line(param_file);
                        
                        split_line(line, tmp_str_items);
                        
                        tmp_int = atom_idx_trip_map[distance(atom_typ_trip_map.begin(), find(atom_typ_trip_map.begin(), atom_typ_trip_map.end(), tmp_str_items[0]))];
                        
                        pair_name[0] = tmp_str_items[1];
                        pair_name[1] = tmp_str_items[2];
                        pair_name[2] = tmp_str_items[3];
                        
                        cutoffval[0] = stod(tmp_str_items[4]);
                        cutoffval[1] = stod(tmp_str_items[5]);
                        cutoffval[2] = stod(tmp_str_items[6]);
                        
                        vector<bool>   disqualified(3,false);
                        
                        chimes_3b_cutoff[tmp_int][0][ get_index_if(trip_params_pair_typs[tmp_int], pair_name[0], disqualified) ] = cutoffval[0];
                        chimes_3b_cutoff[tmp_int][0][ get_index_if(trip_params_pair_typs[tmp_int], pair_name[1], disqualified) ] = cutoffval[1];
                        chimes_3b_cutoff[tmp_int][0][ get_index_if(trip_params_pair_typs[tmp_int], pair_name[2], disqualified) ] = cutoffval[2];
                
                    }
                }
                
                for(int i=0; i<ntrips; i++)
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << i << " " << chimes_3b_cutoff[i][0][0] << " " << chimes_3b_cutoff[i][0][1] << " " << chimes_3b_cutoff[i][0][2] << endl;
            }            
        }
        // With all of the 3 body stuff safely done at this point, we want to flatten
        // chimes_3b_powers by one dimension to make it friendly to the GPU.
        chimes_3b_powers_flat.resize(chimes_3b_powers.size());
        for (int i = 0; i < chimes_3b_powers.size(); i++) {
            for (int j = 0; j < chimes_3b_powers[i].size(); j++) {
                for (int k = 0; k < chimes_3b_powers[i][j].size(); k++) {
                    chimes_3b_powers_flat[i].push_back(chimes_3b_powers[i][j][k]); // should this be j or i?
                }
            }
        }
        
          
    }
    
    // Rewind and read the 4-body Chebyshev pair parameters
    
    if (poly_orders[2] > 0)
    {
        int nquads;
        int tmp_idx;
        
        // Read parameters
        
        param_file.seekg(0);
        
        found_end = false;
    
        while (!found_end)
        {
            line = get_next_line(param_file);
        
			if(line.find("ENDFILE") != string::npos)
                break;    
            
            if(line.find("ATOM PAIR QUADRUPLETS:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                nquads = stoi(tmp_str_items[3]);
                                
                ncoeffs_4b      .resize(nquads);                                 
                chimes_4b_powers.resize(nquads);                                              
                chimes_4b_params.resize(nquads);                       
                chimes_4b_cutoff.resize(nquads);                            
                
                quad_params_atm_chems.resize(nquads);                
                quad_params_pair_typs.resize(nquads);
            }
            
            if(line.find("QUADRUPLETYPE PARAMS:") != string::npos)
            {            
                line = get_next_line(param_file);
                
                split_line(line, tmp_str_items);
                
                tmp_int = stoi(tmp_str_items[1]);
                
                quad_params_atm_chems[tmp_int].push_back(tmp_str_items[3]);
                quad_params_atm_chems[tmp_int].push_back(tmp_str_items[4]);
                quad_params_atm_chems[tmp_int].push_back(tmp_str_items[5]);
                quad_params_atm_chems[tmp_int].push_back(tmp_str_items[6]);

                if (rank == 0)
                    cout << "chimesFF: " << "Read 4B parameters for quadruplets: " << tmp_int << " " << quad_params_atm_chems[tmp_int][0] << " " << quad_params_atm_chems[tmp_int][1] << " " << quad_params_atm_chems[tmp_int][2] << " " << quad_params_atm_chems[tmp_int][3]<< endl;
                
                line = get_next_line(param_file);
                
                split_line(line, tmp_str_items);
            
                quad_params_pair_typs[tmp_int].push_back(tmp_str_items[1]);
                quad_params_pair_typs[tmp_int].push_back(tmp_str_items[2]);
                quad_params_pair_typs[tmp_int].push_back(tmp_str_items[3]);
                quad_params_pair_typs[tmp_int].push_back(tmp_str_items[4]);
                quad_params_pair_typs[tmp_int].push_back(tmp_str_items[5]);
                quad_params_pair_typs[tmp_int].push_back(tmp_str_items[6]);   
		
		// Check for excluded triplet types
	
		if(tmp_str_items[7] != "EXCLUDED:")
		{		                         
	                ncoeffs_4b[tmp_int] = stoi(tmp_str_items[10]);    

	                get_next_line(param_file);
	                get_next_line(param_file);
            
	                vector<int> tmp_int_vec(6);
                
	                for(int i=0; i<ncoeffs_4b[tmp_int]; i++)
	                {                
	                    line = get_next_line(param_file);
	                    split_line(line, tmp_str_items);
                    
	                    tmp_int_vec[0] = stoi(tmp_str_items[1]);
	                    tmp_int_vec[1] = stoi(tmp_str_items[2]);
	                    tmp_int_vec[2] = stoi(tmp_str_items[3]);
	                    tmp_int_vec[3] = stoi(tmp_str_items[4]);
	                    tmp_int_vec[4] = stoi(tmp_str_items[5]);
	                    tmp_int_vec[5] = stoi(tmp_str_items[6]);
                    
	                    chimes_4b_powers[tmp_int].push_back(tmp_int_vec);                 
                    
	                    chimes_4b_params[tmp_int].push_back(stod(tmp_str_items[9]));
                
	                    if (rank == 0)
                        	cout << "chimesFF: " << "\t" << 
                        	chimes_4b_powers[tmp_int][i][0] << " " << 
                	        chimes_4b_powers[tmp_int][i][1] << " " << 
                	        chimes_4b_powers[tmp_int][i][2] << " " << 
                	        chimes_4b_powers[tmp_int][i][3] << " " << 
                	        chimes_4b_powers[tmp_int][i][4] << " " << 
                	        chimes_4b_powers[tmp_int][i][5] << " " <<                                
                	        chimes_4b_params[tmp_int][i] << endl;
                	}
		}
		else
		{
			cout << "chimesFF: \tType is excluded... skipping." << endl;		
		}
            }    
            
            if(line.find("QUADMAPS:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                n_quad_maps = stoi(tmp_str_items[1]);
                
                atom_idx_quad_map.resize(n_quad_maps);
                atom_typ_quad_map.resize(n_quad_maps);
                    
                if (rank == 0)            
                    cout << "chimesFF: " << "Built the following 4-body pair \"slow\" map:" << endl;
            
                for(int i=0; i<n_quad_maps; i++)
                {
                    line = get_next_line(param_file);
                    split_line(line, tmp_str_items);
                
                    atom_idx_quad_map[i] = stoi(tmp_str_items[0]);
                    atom_typ_quad_map[i] =      tmp_str_items[1];
                
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << atom_idx_quad_map[i] << " " << atom_typ_quad_map[i] << endl;
                }        
                
                if (rank == 0)
                    cout << "chimesFF: " << "Built the following 4-body pair \"fast\" map:" << endl;

                atom_int_quad_map.resize(natmtyps*natmtyps*natmtyps*natmtyps);
                
                for(int i=0; i<natmtyps; i++)
                {
                    for (int j=0; j<natmtyps; j++)
                    {
                        for(int k=0; k<natmtyps; k++)
                        {
                            for(int l=0; l<natmtyps; l++)
                            {                            
                                // Get the quad type name for the set of atoms
                            
                                tmp_str = "";
                                
                                
                                tmp_int  = get_proper_pair(atmtyps[i], atmtyps[j]);
                                tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];    

                                tmp_int  = get_proper_pair(atmtyps[i], atmtyps[k]);
                                tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];    
                
                                tmp_int  = get_proper_pair(atmtyps[i], atmtyps[l]);
                                tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];    
                            
                                tmp_int  = get_proper_pair(atmtyps[j], atmtyps[k]);
                                tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];    
                            
                                tmp_int  = get_proper_pair(atmtyps[j], atmtyps[l]);
                                tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];
                            
                                tmp_int  = get_proper_pair(atmtyps[k], atmtyps[l]);
                                tmp_str += pair_params_atm_chem_1[tmp_int] + pair_params_atm_chem_2[tmp_int];                                                                                                

                                tmp_int = get_index(atom_typ_quad_map, tmp_str);
                            
                                tmp_idx = i*natmtyps*natmtyps*natmtyps + j*natmtyps*natmtyps + k*natmtyps + l;

                                atom_int_quad_map[ tmp_idx ] = atom_idx_quad_map[tmp_int];

                                if (rank == 0)
                                    cout << "chimesFF: " << "\t" << tmp_idx << " " << atom_int_quad_map[ tmp_idx  ]  << endl;
                            }
                        }
                    }
                }
            }            
        }
        
        // Set up cutoffs ... First set to match 2-body, then read special if they exist
        
        int atmtyp_1,  atmtyp_2,  atmtyp_3,  atmtyp_4;
        int pairtyp_1, pairtyp_2, pairtyp_3, pairtyp_4, pairtyp_5, pairtyp_6;
        
        for(int i=0; i<nquads; i++) 
        {
            // Figure out the atom type index for each atom in the quadruplet type 
                        
            atmtyp_1 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), quad_params_atm_chems[i][0]));    
            atmtyp_2 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), quad_params_atm_chems[i][1]));    
            atmtyp_3 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), quad_params_atm_chems[i][2]));    
            atmtyp_4 = distance(atmtyps.begin(), find(atmtyps.begin(), atmtyps.end(), quad_params_atm_chems[i][3]));    
                        
            // Figure out the corresponding 2-body pair type
            
            pairtyp_1 = atom_int_pair_map[ atmtyp_1*natmtyps + atmtyp_2 ];
            pairtyp_2 = atom_int_pair_map[ atmtyp_1*natmtyps + atmtyp_3 ];
            pairtyp_3 = atom_int_pair_map[ atmtyp_1*natmtyps + atmtyp_4 ];
            pairtyp_4 = atom_int_pair_map[ atmtyp_2*natmtyps + atmtyp_3 ];
            pairtyp_5 = atom_int_pair_map[ atmtyp_2*natmtyps + atmtyp_4 ];
            pairtyp_6 = atom_int_pair_map[ atmtyp_3*natmtyps + atmtyp_4 ];            
    
            // Set the default inner/outer cutoffs to the corresponding 2-body value                    

            chimes_4b_cutoff[i].resize(2);            

            chimes_4b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_1][0]);
            chimes_4b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_2][0]);
            chimes_4b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_3][0]);
            chimes_4b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_4][0]);
            chimes_4b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_5][0]);
            chimes_4b_cutoff[i][0].push_back(chimes_2b_cutoff[pairtyp_6][0]);              
            
            chimes_4b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_1][1]);
            chimes_4b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_2][1]);
            chimes_4b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_3][1]);          
            chimes_4b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_4][1]);
            chimes_4b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_5][1]);
            chimes_4b_cutoff[i][1].push_back(chimes_2b_cutoff[pairtyp_6][1]);                                              
        }
        
        param_file.seekg(0);
        
        int    nentries;
        double cutval;
        
        found_end = false;
        
        while (!found_end)
        {
			line = get_next_line(param_file);
        
			if(line.find("ENDFILE") != string::npos)
                break;                
            
            if(line.find("SPECIAL 4B S_MAXIM:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                if (rank == 0)
                    cout << "chimesFF: " << "Set the following special 4-body outer cutoffs: " << endl;
                
                if(tmp_str_items[3] == "ALL")
                {
                    cutval = stod(tmp_str_items[4]);
                                        
                    for(int i=0; i<nquads; i++)
                    {                
                        chimes_4b_cutoff[i][1][0] = cutval;
                        chimes_4b_cutoff[i][1][1] = cutval;
                        chimes_4b_cutoff[i][1][2] = cutval;
                        chimes_4b_cutoff[i][1][3] = cutval;
                        chimes_4b_cutoff[i][1][4] = cutval;
                        chimes_4b_cutoff[i][1][5] = cutval;                                                      
                    }
                }
                else
                {
                    nentries = stoi(tmp_str_items[4]);
                    
                    vector<string> pair_name(6);
                    vector<double> cutoffval(6);

                    for(int i=0; i<nentries; i++)
                    {
                        line = get_next_line(param_file);
                        
                        split_line(line, tmp_str_items);
                        
                        tmp_int = atom_idx_quad_map[distance(atom_typ_quad_map.begin(), find(atom_typ_quad_map.begin(), atom_typ_quad_map.end(), tmp_str_items[0]))];

                        pair_name[0] = tmp_str_items[1];
                        pair_name[1] = tmp_str_items[2];
                        pair_name[2] = tmp_str_items[3];
                        pair_name[3] = tmp_str_items[4];
                        pair_name[4] = tmp_str_items[5];
                        pair_name[5] = tmp_str_items[6];
                        
                        cutoffval[0] = stod(tmp_str_items[7 ]);
                        cutoffval[1] = stod(tmp_str_items[8 ]);
                        cutoffval[2] = stod(tmp_str_items[9 ]);
                        cutoffval[3] = stod(tmp_str_items[10]);
                        cutoffval[4] = stod(tmp_str_items[11]);
                        cutoffval[5] = stod(tmp_str_items[12]);
                        
                        vector<bool>   disqualified(6,false);
                        
                        chimes_4b_cutoff[tmp_int][1][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[0], disqualified) ] = cutoffval[0];
                        chimes_4b_cutoff[tmp_int][1][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[1], disqualified) ] = cutoffval[1];
                        chimes_4b_cutoff[tmp_int][1][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[2], disqualified) ] = cutoffval[2];    
                        chimes_4b_cutoff[tmp_int][1][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[3], disqualified) ] = cutoffval[3];
                        chimes_4b_cutoff[tmp_int][1][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[4], disqualified) ] = cutoffval[4];
                        chimes_4b_cutoff[tmp_int][1][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[5], disqualified) ] = cutoffval[5];
					}
                }
                
                for(int i=0; i<nquads; i++)
                {                
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << i << " " 
                        << chimes_4b_cutoff[i][1][0] << " " 
                        << chimes_4b_cutoff[i][1][1] << " " 
                        << chimes_4b_cutoff[i][1][2] << " " 
                        << chimes_4b_cutoff[i][1][3] << " " 
                        << chimes_4b_cutoff[i][1][4] << " " 
                        << chimes_4b_cutoff[i][1][5] << endl;
                }                
            }

            if(line.find("SPECIAL 4B S_MINIM:") != string::npos)
            {
                split_line(line, tmp_str_items);
                
                if (rank == 0)
                    cout << "chimesFF: " << "Set the following special 4-body inner cutoffs: " << endl;
                
                if(tmp_str_items[3] == "ALL")
                {
                    cutval = stod(tmp_str_items[4]);
                    
                    for(int i=0; i<nquads; i++)
                    {                
                        chimes_4b_cutoff[i][0][0] = cutval;
                        chimes_4b_cutoff[i][0][1] = cutval;
                        chimes_4b_cutoff[i][0][2] = cutval;
                        chimes_4b_cutoff[i][0][3] = cutval;
                        chimes_4b_cutoff[i][0][4] = cutval;
                        chimes_4b_cutoff[i][0][5] = cutval;                         
                    }
                }
                else
                {
                    nentries = stoi(tmp_str_items[4]);
                    
                    vector<string> pair_name(6);
                    vector<double> cutoffval(6);

					for(int i=0; i<nquads; i++)
					{
						chimes_4b_cutoff[i][0][0] = -1.0;
						chimes_4b_cutoff[i][0][1] = -1.0;
						chimes_4b_cutoff[i][0][2] = -1.0;
						chimes_4b_cutoff[i][0][3] = -1.0;
						chimes_4b_cutoff[i][0][4] = -1.0;
						chimes_4b_cutoff[i][0][5] = -1.0;
					}

                    for(int i=0; i<nentries; i++)
                    {
                        line = get_next_line(param_file);
                        
                        split_line(line, tmp_str_items);
                        
                        tmp_int = atom_idx_quad_map[distance(atom_typ_quad_map.begin(), find(atom_typ_quad_map.begin(), atom_typ_quad_map.end(), tmp_str_items[0]))];

                        pair_name[0] = tmp_str_items[1];
                        pair_name[1] = tmp_str_items[2];
                        pair_name[2] = tmp_str_items[3];
                        pair_name[3] = tmp_str_items[4];
                        pair_name[4] = tmp_str_items[5];
                        pair_name[5] = tmp_str_items[6];
                        
                        cutoffval[0] = stod(tmp_str_items[7 ]);
                        cutoffval[1] = stod(tmp_str_items[8 ]);
                        cutoffval[2] = stod(tmp_str_items[9 ]);
                        cutoffval[3] = stod(tmp_str_items[10]);
                        cutoffval[4] = stod(tmp_str_items[11]);
                        cutoffval[5] = stod(tmp_str_items[12]);
                        
                        vector<bool>   disqualified(6,false);
                        
                        chimes_4b_cutoff[tmp_int][0][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[0], disqualified) ] = cutoffval[0];
                        chimes_4b_cutoff[tmp_int][0][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[1], disqualified) ] = cutoffval[1];
                        chimes_4b_cutoff[tmp_int][0][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[2], disqualified) ] = cutoffval[2];    
                        chimes_4b_cutoff[tmp_int][0][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[3], disqualified) ] = cutoffval[3];
                        chimes_4b_cutoff[tmp_int][0][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[4], disqualified) ] = cutoffval[4];
                        chimes_4b_cutoff[tmp_int][0][ get_index_if(quad_params_pair_typs[tmp_int], pair_name[5], disqualified) ] = cutoffval[5];                     
                    }
                }
                
                for(int i=0; i<nquads; i++)
                {                
                    if (rank == 0)
                        cout << "chimesFF: " << "\t" << i << " " 
                        << chimes_4b_cutoff[i][1][0] << " " 
                        << chimes_4b_cutoff[i][1][1] << " " 
                        << chimes_4b_cutoff[i][1][2] << " " 
                        << chimes_4b_cutoff[i][1][3] << " " 
                        << chimes_4b_cutoff[i][1][4] << " " 
                        << chimes_4b_cutoff[i][1][5] << endl;
                }                
            }            
        }

        //set up flat version of powers for 4 body.
        chimes_4b_powers_flat.resize(chimes_4b_powers.size());
        for (int i = 0; i < chimes_4b_powers.size(); i++) {
            for (int j = 0; j < chimes_4b_powers[i].size(); j++) {
                for (int k = 0; k < chimes_4b_powers[i][j].size(); k++) {
                    chimes_4b_powers_flat[i].push_back(chimes_4b_powers[i][j][k]);
                }
            }
        }    
    }
    
    param_file.close();    
}

void chimesFF::set_polys_out_of_range(vector<double> &Tn, vector<double> &Tnd, double dx, double x, int poly_order, double inner_cutoff, double exprlen, double dx_dr)
{
    //  Sets the value of the Chebyshev polynomials (Tn) and their derivatives (Tnd) when dx is < inner_cutoff.
    //  Tnd is the derivative with respect to the interatomic distance, not the transformed distance (x).
    //	
    //  The derivative Tnd is continuously set to zero inside the cutoff.
    //  The exponential smoothing distance is set to ChimesFF::inner_smooth_distance.
    //  x, exprlen, and dx_dr are evaluated at the inner cutoff.
    //	
    //  dx is the pair distance, which is assumed to be less than inner_cutoff.
    Tn[0] = 1.0;
    Tn[1] = x;

    // Start the derivative setup. Set the first two 1st-kind Cheby's equal to the first two of the 2nd-kind

    Tnd[0] = 1.0;
    Tnd[1] = 2.0 * x;
    
    // Use recursion to set up the higher n-value Tn and Tnd's
    for ( int i = 2; i <= poly_order; i++ ) 
    {
        Tn[i]  = 2.0 * x *  Tn[i-1] -  Tn[i-2];
        Tnd[i] = 2.0 * x * Tnd[i-1] - Tnd[i-2];
    }
    
    // Now multiply by n to convert Tnd's to actual derivatives of Tn

    for ( int i = poly_order; i >= 1; i-- ) 
        Tnd[i] = i * dx_dr * Tnd[i-1];

    Tnd[0] = 0.0;

    // Exponential damping of the derivative.
    double damp_fac = exp( (dx-inner_cutoff) / inner_smooth_distance ) ;
      
    // Correct Tn outside of the range using the damping factor.
    for ( int i = 0 ; i <= poly_order ; i++ )
    {
        Tn[i]  += inner_smooth_distance * (damp_fac-1.0)  * Tnd[i] ;
        Tnd[i] *= damp_fac ;
    }     
}

inline double chimesFF::dr2_3B(const double *dr2, int i, int j, int k, int l)
{
    // Access the dr2 distance tensor for a 3 body interaction.
    return(dr2[i*CHDIM*3*CHDIM + j*3*CHDIM + k*CHDIM + l]) ;
}

inline double chimesFF::dr2_4B(const double *dr2, int i, int j, int k, int l)
{
    // Access the dr2 distance tensor for a 4 body interaction.
    return(dr2[i*CHDIM*6*CHDIM + j*6*CHDIM + k*CHDIM + l]) ;
}

inline void chimesFF::init_distance_tensor(double *dr2, const vector<double> & dr, int npairs)
{
    for ( int i = 0 ; i < npairs ; i++ )
        for ( int j = 0 ; j < CHDIM ; j++ )
            for ( int k = 0 ; k < npairs ; k++ )
                for ( int l = 0 ; l < CHDIM ; l++ )
                    dr2[i* CHDIM * npairs * CHDIM + j * npairs * CHDIM + k * CHDIM + l] = dr[i*CHDIM+j] * dr[k*CHDIM+l] ;
}

void chimesFF::compute_1B(const int typ_idx, double & energy )
{
    // Compute 1b (input: a single atom type index... outputs (updates) energy

    energy += energy_offsets[typ_idx];
}



// Overload for calls from LAMMPS                 
void chimesFF::compute_2B(const double dx, const vector<double> & dr, const vector<int> typ_idxs, vector<double> & force, vector<double> & stress, double & energy, chimes2BTmp &tmp)
{              
    double dummy_force_scalar;
    compute_2B(dx, dr, typ_idxs, force, stress, energy, tmp, dummy_force_scalar);                                                               
}
void chimesFF::compute_2B(const double dx, const vector<double> & dr, const vector<int> typ_idxs, vector<double> & force, vector<double> & stress, double & energy, chimes2BTmp &tmp, double & force_scalar_in)
{
    // Compute 2b (input: 2 atoms or distances, corresponding types... outputs (updates) force, acceleration, energy, stress
    //
    // Input parameters:
    //
    // dx: Scalar (pair distance)
    // dr: 1d-Array (pair distance: [x, y, and z-component]) 
    // Force: [natoms in interaction set][x,y, and z-component] *note
    // Stress [sxx, sxy, sxz, syy, syz, szz]  *note
    // Energy: Scalar; energy for interaction set
    // Tmp: Temporary storage for calculation.
    
    // Assumes atom indices start from zero
    // Assumes distances are atom_2 - atom_1
    //
    // *note: force is a packed array of coordinates.

    int     pair_idx;    
    double  fcut;
    double  fcutderiv;

    // tmp.resize(poly_orders[0]+1) ;
    
    // Use references for readability.
    vector<double> &Tn = tmp.Tn ;
    vector<double> &Tnd = tmp.Tnd ;
    
    pair_idx = atom_int_pair_map[ typ_idxs[0]*natmtyps + typ_idxs[1] ];

    if (dx >= chimes_2b_cutoff[pair_idx][1])
        return;    

    set_cheby_polys(Tn, Tnd, dx, pair_idx, chimes_2b_cutoff[pair_idx][0], chimes_2b_cutoff[pair_idx][1], 0);
    
    get_fcut(dx, chimes_2b_cutoff[pair_idx][1], fcut, fcutderiv);

    double dx_inv = ( dx > 0.0 ) ? 1.0 / dx : 1e20 ;

    // need to create some C-style arrays/pointers to feed into GPU
    // and receive results.

    // Host and GPU pointers - these all need to be freed
    // at the end.

    // pointers to get force and stress data back at the end of calculations
    double *host_stress, *host_force;
    // device/GPU pointers for the items being transferred over.
    double *device_chimes_params, *device_Tn, *device_Tnd, *device_force, *device_stress;
    int *device_chimes_pows;
    // GPU memory allocation

    // Start with copying dr to constant memory - it doesnt
    // change for the duration of the call to the GPU and it 
    // is used by every single thread.
   
    hipMemcpyToSymbol(HIP_SYMBOL(dr_gpu), dr.data(), CHDIM*sizeof(double));
    

    //set energy to whatever the input value should be
    //ok this finally transfers properly but I think if I want
    //to use this for a multi threaded application I need to switch this to
    //https://stackoverflow.com/a/2620144
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_energy), &energy, sizeof(double));

    // allocate GPU memory for device pointers

    hipMalloc(&device_chimes_params, chimes_2b_params[pair_idx].size() * sizeof(double));
    hipMalloc(&device_Tn, Tn.size() * sizeof(double));
    hipMalloc(&device_Tnd, Tnd.size() * sizeof(double));
    hipMalloc(&device_force, force.size() * sizeof(double));
    hipMalloc(&device_stress, stress.size() * sizeof(double));
    hipMalloc(&device_chimes_pows, chimes_2b_pows[pair_idx].size() * sizeof(int));

    // Transfer memory from host to GPU.

    hipMemcpy(device_chimes_params, chimes_2b_params[pair_idx].data(), chimes_2b_params[pair_idx].size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_Tn, Tn.data(), Tn.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_Tnd, Tnd.data(), Tnd.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_force, force.data(), force.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_stress, stress.data(), stress.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_chimes_pows, chimes_2b_pows[pair_idx].data(), chimes_2b_pows[pair_idx].size() * sizeof(int), hipMemcpyHostToDevice);

    //grid and block dimension setup - for controlling threads on the gpu.

    int blockSize = 512;  // start with this - can consider other options depending on some of the other
    // strategies I end up using.
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(ceil((double)ncoeffs_2b[pair_idx]/blockSize), 1 , 1);

    // Starting the kernel - houston, we are cleared for liftoff
    compute2B_helper<<<dimGrid, dimBlock>>>(ncoeffs_2b[pair_idx], fcut, fcutderiv, dx_inv, device_chimes_params,
    device_chimes_pows, device_Tn, device_Tnd, device_force, device_stress);
    hipDeviceSynchronize();  // do not proceed on the CPU until the GPU has finished its calculations.
    // kernel completion - update host memory with results
   
    //allocate memory for our host pointers to receive the newly updated data
    //TODO - I'm not convinced that this is the best way to handle this, maybe return to this
    //in the future when I'm wiser and I'm older

    host_stress = (double *)malloc(stress.size() * sizeof(double));
    host_force = (double *)malloc(force.size() * sizeof(double));

    //transfer new force and stress values from the GPU to the CPU
    //perhaps try putting stress.data() in the recieving spot and seeing what happens?  I've never
    //really tried that before.
    hipMemcpy(host_stress, device_stress, stress.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(host_force, device_force, force.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&energy, HIP_SYMBOL(gpu_energy), sizeof(double));
    

    //update the storage vectors on the host - replace old data with new data from GPU calculations.
    stress = std::vector<double>(host_stress, host_stress + stress.size());
    force = std::vector<double>(host_force, host_force + force.size());


    // just need to solve getting energy - 
    // https://stackoverflow.com/questions/2619296/how-to-return-a-single-variable-from-a-cuda-kernel-function
    // answers in this thread may be helpful

    // free memory used - be very careful not to cause
    // memory leaks since this will almost certainly be called
    // multiple times by host programs - memory leaks
    // will very rapidly cause out of memory errors.

    //clean up time!  Free takes care of malloc memory, hipFree takes care
    //of hipMalloc memory on the GPU.
    free(host_stress);
    free(host_force);
    hipFree(device_chimes_params);
    hipFree(device_Tn);
    hipFree(device_Tnd);
    hipFree(device_chimes_pows);
    hipFree(device_force);
    hipFree(device_stress);
    
    /*
    for(int coeffs=0; coeffs<ncoeffs_2b[pair_idx]; coeffs++)
    {
        double coeff_val = [coeffs];        
        
        energy += coeff_val * fcut * Tn[ chimes_2b_pows[pair_idx][coeffs]+1 ];
                                                
        double deriv = fcut * Tnd[ chimes_2b_pows[pair_idx][coeffs]+1 ]  + fcutderiv * Tn[ chimes_2b_pows[pair_idx][coeffs]+1 ];    

        double force_scalar = coeff_val * deriv * dx_inv ; 

        force[0*CHDIM+0] += force_scalar * dr[0];
        force[0*CHDIM+1] += force_scalar * dr[1];
        force[0*CHDIM+2] += force_scalar * dr[2];
        
        force[1*CHDIM+0] -= force_scalar * dr[0];
        force[1*CHDIM+1] -= force_scalar * dr[1];
        force[1*CHDIM+2] -= force_scalar * dr[2];
        
        // xx xy xz yy yz zz
        // 0  1  2  3  4  5
        
        // xx xy xz yx yy yz zx zy zz
        // 0  1  2  3  4  5  6  7  8
        // *           *           *
        
        stress[0] -= force_scalar * dr[0] * dr[0]; // xx tensor component
        stress[1] -= force_scalar * dr[0] * dr[1]; // xy tensor component 
        stress[2] -= force_scalar * dr[0] * dr[2]; // xz tensor component
        stress[3] -= force_scalar * dr[1] * dr[1]; // yy tensor component
        stress[4] -= force_scalar * dr[1] * dr[2]; // yz tensor component
        stress[5] -= force_scalar * dr[2] * dr[2]; // zz tensor component
            
    } */


    double E_penalty = 0.0 ;
    double force_scalar ;
    get_penalty(dx, pair_idx, E_penalty , force_scalar); 

    if ( E_penalty > 0.0 ) 
    {
        energy += E_penalty;

        force_scalar /= dx ;

        // Note: force_scalar is negative (LEF) 7/30/21.
        force[0*CHDIM+0] += force_scalar * dr[0];
        force[0*CHDIM+1] += force_scalar * dr[1];
        force[0*CHDIM+2] += force_scalar * dr[2];
        
        force[1*CHDIM+0] -= force_scalar * dr[0];
        force[1*CHDIM+1] -= force_scalar * dr[1];
        force[1*CHDIM+2] -= force_scalar * dr[2];

        // Update stress according to penalty force. (LEF) 07/30/21
        stress[0] -= force_scalar  * dr[0] * dr[0]; // xx tensor component
        stress[1] -= force_scalar  * dr[0] * dr[1]; // xy tensor component 
        stress[2] -= force_scalar  * dr[0] * dr[2]; // xz tensor component
        stress[3] -= force_scalar  * dr[1] * dr[1]; // yy tensor component
        stress[4] -= force_scalar  * dr[1] * dr[2]; // yz tensor component
        stress[5] -= force_scalar  * dr[2] * dr[2]; // zz tensor component

    }
    
    force_scalar_in = force_scalar;
}

// Overload for calls from LAMMPS  
void chimesFF::compute_3B(const vector<double> & dx, const vector<double> & dr, const vector<int> & typ_idxs, vector<double> & force, vector<double> & stress, double & energy, chimes3BTmp &tmp)
{
	vector<double> dummy_force_scalar(3);
	compute_3B(dx, dr, typ_idxs, force, stress, energy, tmp, dummy_force_scalar);
}
void chimesFF::compute_3B(const vector<double> & dx, const vector<double> & dr, const vector<int> & typ_idxs, vector<double> & force, vector<double> & stress, double & energy, chimes3BTmp &tmp, vector<double> & force_scalar_in)
{
    // Compute 3b (input: 3 atoms or distances, corresponding types... outputs (updates) force, acceleration, energy, stress
    //
    // Input parameters:
    //
    // dx_ij: Scalar (pair distance)
    // dr_ij: 1d-Array (pair distance: [x, y, and z-component])
    // Force: [natoms in interaction set][x,y, and z-component] *note
    // Stress [sxx, sxy, sxz, syy, syz, szz] 
    // Energy: Scalar; energy for interaction set
    // Tmp: Temporary storage for 3-body interactions.
    
    // Assumes atom indices start from zero
    // Assumes distances are atom_2 - atom_1
    //
    // *note: force and dr are packed vectors of coordinates.
    
    const int natoms = 3;                   // Number of atoms in an interaction set
    const int npairs = natoms*(natoms-1)/2; // Number of pairs in an interaction set
    
    // tmp.resize(poly_orders[1]) ;
    
    vector<double> &Tn_ij  = tmp.Tn_ij ;
    vector<double> &Tn_ik  = tmp.Tn_ik ;
    vector<double> &Tn_jk  = tmp.Tn_jk ;   // The Chebyshev polymonials
    vector<double> &Tnd_ij = tmp.Tnd_ij ;
    vector<double> &Tnd_ik = tmp.Tnd_ik ;
    vector<double> &Tnd_jk = tmp.Tnd_jk ;  // The Chebyshev polymonial derivatives

    //I think I want to store all of these device pointers in a struct of some sort, especially for four body, to cut
    //down on the number of arguments I have to pass, especially since there is a limit of 32 pointers/256 bytes I think.

    // Avoid allocating std::vector quantities.  Heap memory allocation is slow on the GPU.
    // fixed-length C arrays are allocated on the stack.
    double fcut[npairs] ;
    double fcutderiv[npairs] ;
    //double deriv[npairs];

    

#if DEBUG == 1  
    if ( dr.size() != 9 )
    {
        cout << "Error: dr should have length = 9.  Current length = " << dr.size() << endl ;
        exit(0) ;
    }
#endif


    int type_idx =  typ_idxs[0]*natmtyps*natmtyps + typ_idxs[1]*natmtyps + typ_idxs[2] ;
    int tripidx = atom_int_trip_map[type_idx];

    if(tripidx < 0)    // Skipping an excluded interaction
        return;
    
    // Check whether cutoffs are within allowed ranges
    vector<int> & mapped_pair_idx = pair_int_trip_map[type_idx] ;
        
    if (dx[0] >= chimes_3b_cutoff[ tripidx ][1][mapped_pair_idx[0]])    // ij
        return;    
    if (dx[1] >= chimes_3b_cutoff[ tripidx ][1][mapped_pair_idx[1]])    // ik
        return;    
    if (dx[2] >= chimes_3b_cutoff[ tripidx ][1][mapped_pair_idx[2]])    // jk
        return;    
    
    // At this point, all distances are within allowed ranges. We can now proceed to the force/stress/energy calculation

#ifdef USE_DISTANCE_TENSOR  
    // Tensor product of displacement vectors.
    double dr2[CHDIM*CHDIM*npairs*npairs] ;
    init_distance_tensor(dr2, dr, npairs) ;
#endif

    // Set up the polynomials

    set_cheby_polys(Tn_ij, Tnd_ij, dx[0], atom_int_pair_map[ typ_idxs[0]*natmtyps + typ_idxs[1] ], chimes_3b_cutoff[tripidx][0][mapped_pair_idx[0]], chimes_3b_cutoff[tripidx][1][mapped_pair_idx[0]], 1);
    set_cheby_polys(Tn_ik, Tnd_ik, dx[1], atom_int_pair_map[ typ_idxs[0]*natmtyps + typ_idxs[2] ], chimes_3b_cutoff[tripidx][0][mapped_pair_idx[1]], chimes_3b_cutoff[tripidx][1][mapped_pair_idx[1]], 1);
    set_cheby_polys(Tn_jk, Tnd_jk, dx[2], atom_int_pair_map[ typ_idxs[1]*natmtyps + typ_idxs[2] ], chimes_3b_cutoff[tripidx][0][mapped_pair_idx[2]], chimes_3b_cutoff[tripidx][1][mapped_pair_idx[2]], 1);
    
    
    // Set up the smoothing functions
        
    get_fcut(dx[0], chimes_3b_cutoff[tripidx][1][mapped_pair_idx[0]], fcut[0], fcutderiv[0]);
    get_fcut(dx[1], chimes_3b_cutoff[tripidx][1][mapped_pair_idx[1]], fcut[1], fcutderiv[1]);
    get_fcut(dx[2], chimes_3b_cutoff[tripidx][1][mapped_pair_idx[2]], fcut[2], fcutderiv[2]);
    double fcut_all =  fcut[0] * fcut[1] * fcut[2] ;

    // Product of 2 fcuts divided by dx. Index i = product of all fcuts except i.
    double fcut_2[npairs] ;
    fcut_2[0] = fcut[1] * fcut[2] / dx[0] ;
    fcut_2[1] = fcut[0] * fcut[2] / dx[1] ;
    fcut_2[2] = fcut[0] * fcut[1] / dx[2] ;

    // Start the force/stress/energy calculation
    //double coeff;
    //int powers[npairs] ;
    //double force_scalar[npairs] ;

    // GPU IMPLEMENTATION

    // Let the book keeping begin :(
    
    // What needs to go to the GPU?
    // ncoeffs_3b[tripidx] can be passed in the kernel call.
    // chimes_3b_params[tripidx] needs memcpy
    // chimes_3b_powers[tripidx] needs memcpy and HAS TO BE FLATTENED :(
    // flattening this every time may be expensive?  Talk with Professor Lindsey about where else
    // the chimes_3b_powers is used in the code and if I can set it up to be flat initally.
    // energy has to be either the __device__ energy that I have been using or it
    // just needs to be an allocated double, which is what I will have to do if I want it to be MPI capable.
    // fcut_all can be passed in the kernel call.
    // fcut array should be sent to constant memory
    // fcutderiv array should also be sent to constant memory.
    // fcut2 should also be sent to constant memory.
    // Wait - I'm not sure how constant memory interacts with MPI calls.
    // I think it may be fine since at least on the CPU each MPI thing is a
    // copy and has their own copies of the variables/their own context?
    // thats a problem for later, its fixable if I am wrong.
    // dr and dr2_3b should both be sent to constant memory.


    // need to create some C-style arrays/pointers to feed into GPU
    // and receive results.
    // Host and GPU pointers - these all need to be freed
    // at the end.

    // host pointers to update stress and force at the end.
    // double *host_stress, *host_force;
    // device/GPU pointers for the items being transferred over.
    //poly_pointers_3b *device_polys; // I think the whole struct now has to
    // be copied over to memory, increasing memory accesses?
    // Maybe more arguments is the answer?
    // If I do it this way, I pass a pointer to a struct.
    // I then have a memory access to the struct to get the pointers in the struct.
    // Or I can pass all of those pointers to the kernel call (assuming they fit)
    // and they should end up as part of the registers for each thread.
    // However, that's a lot of arguments... which is usually considered
    // to be poor practice for other languages at least.  Hmm
    // Its not that big so I think I start with trying to pass by value
    // cause looking at the profiling from the two body version the
    // hipMalloc and free calls are very expensive.

    double *device_chimes_params, *device_force, *device_stress;
    int *device_chimes_pows; // need this but would like it to be flattened
    // before the compute call if possible.
    
    //device_polys = (poly_pointers_3b *)malloc(sizeof(poly_pointers_3b));
    struct poly_pointers_3b device_polys;
    // Allocate GPU memory for device pointers, including the ones in device_polys.
    // Ouch this is a lot of malloc calls.
    // Could put all of these in a struct and then copy the struct to the gpu instead
    // of sending a struct with pointers to each one - that would cut down on hipMalloc calls,
    // but will mean alot of malloc calls on the cpu - I have to think those would be faster
    // though?
    // Lets go have lunch and think about this, I dont like making decisions
    // while hungry.

    hipMalloc(&device_chimes_params, chimes_3b_params[tripidx].size() * sizeof(double));
    hipMalloc(&device_force, force.size() * sizeof(double));
    hipMalloc(&device_stress, stress.size() * sizeof(double));
    hipMalloc(&device_chimes_pows, chimes_3b_powers_flat[tripidx].size() * sizeof(double));
    // mallocs all done except for powers, which I want to flatten first.

    // Tn and Tnd mallocs
    hipMalloc(&device_polys.Tn_ij, Tn_ij.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_ik, Tn_ik.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_jk, Tn_jk.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_ij, Tnd_ij.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_ik, Tnd_ik.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_jk, Tnd_jk.size() * sizeof(double));

    // Begin transferring memory over to the GPU.

    // Start off with the stuff going over to constant memory.
    if (hipMemcpyToSymbol(HIP_SYMBOL(fcut_3b), fcut, npairs * sizeof(double)) != hipSuccess) {
        std::cout << "An error occured at line 1810" << std::endl;
        exit(1);
    }

    if (hipMemcpyToSymbol(HIP_SYMBOL(fcutderiv_3b), fcutderiv, npairs * sizeof(double)) != hipSuccess) {
        std::cout << "An error occured at line 1815" << std::endl;
        exit(1);
    }

    if (hipMemcpyToSymbol(HIP_SYMBOL(fcut2_3b), fcut_2, npairs * sizeof(double)) != hipSuccess) {
        std::cout << "An error occured at line 1820" << std::endl;
        exit(1);
    }
    
    if (hipMemcpyToSymbol(HIP_SYMBOL(dr_3b), dr.data(), dr.size() * sizeof(double)) != hipSuccess) {
        std::cout << "An error occured at line 1825" << std::endl;
        exit(1);
    }

    if (hipMemcpyToSymbol(HIP_SYMBOL(pair_idx_3b), mapped_pair_idx.data(), mapped_pair_idx.size() * sizeof(int)) != hipSuccess) {
        std::cout << "An error occured at line 1830" << std::endl;
        exit(1);
    }
    
    
    
    
    #ifdef USE_DISTANCE_TENSOR
        if (hipMemcpyToSymbol(HIP_SYMBOL(dr2_3b), dr2, sizeof(dr2_3b)) != hipSuccess) {
            std::cout << "An error occured at line 1842" << std::endl;
            exit(1);
        }
        
    #endif
    if (hipMemcpyToSymbol(HIP_SYMBOL(gpu_energy), &energy, sizeof(double)) != hipSuccess) {
        std::cout << "An error occured at line 1848" << std::endl;
        exit(1);
    }
    

    // constant memory stuff is completed, time for dynamically allocated stuff

    hipMemcpy(device_chimes_params, chimes_3b_params[tripidx].data(), chimes_3b_params[tripidx].size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_force, force.data(), force.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_stress, stress.data(), stress.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_chimes_pows, chimes_3b_powers_flat[tripidx].data(), chimes_3b_powers_flat[tripidx].size() * sizeof(int), hipMemcpyHostToDevice);

    //memcpy for the Tn and Tnds
    hipMemcpy(device_polys.Tn_ij, Tn_ij.data(), Tn_ij.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_ik, Tn_ik.data(), Tn_ik.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_jk, Tn_jk.data(), Tn_jk.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_ij, Tnd_ij.data(), Tnd_ij.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_ik, Tnd_ik.data(), Tnd_ik.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_jk, Tnd_jk.data(), Tnd_jk.size() * sizeof(double), hipMemcpyHostToDevice);


    // Memory transfer should now be done - time to do the grid and block dimension setup
    int blockSize = 512;
    // Can start with this
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(ceil((double)ncoeffs_3b[tripidx]/blockSize), 1 , 1);

    // Start the kernel.
    compute3b_helper<<<dimGrid, dimBlock>>>(ncoeffs_3b[tripidx], fcut_all, device_chimes_params, device_chimes_pows, device_force, device_stress, device_polys);
    hipDeviceSynchronize();

    // kernel is now finished at this point - time to get the results we
    // need from the GPU.

    hipMemcpyFromSymbol(&energy, HIP_SYMBOL(gpu_energy), sizeof(double));
    hipMemcpy(stress.data(), device_stress, stress.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(force.data(), device_force, force.size() * sizeof(double), hipMemcpyDeviceToHost);
    // testing this.

    // memory clean up
    hipFree(device_chimes_params);
    hipFree(device_chimes_pows);
    hipFree(device_force);
    hipFree(device_stress);
    hipFree(device_polys.Tn_ij);
    hipFree(device_polys.Tn_ik);
    hipFree(device_polys.Tn_jk);
    hipFree(device_polys.Tnd_ij);
    hipFree(device_polys.Tnd_ik);
    hipFree(device_polys.Tnd_jk);
    //free(device_polys);
    
    /*
    for(int coeffs=0; coeffs<ncoeffs_3b[tripidx]; coeffs++)
    {
        coeff = chimes_3b_params[tripidx][coeffs];
        
        powers[0] = chimes_3b_powers[tripidx][coeffs][mapped_pair_idx[0]];
        powers[1] = chimes_3b_powers[tripidx][coeffs][mapped_pair_idx[1]];
        powers[2] = chimes_3b_powers[tripidx][coeffs][mapped_pair_idx[2]];

        // For powers - the mapped pair will always be a size of 3,
        // so I can flatten by that dimension and then index in with coeffs * 3 + mapped_pair_index

        //GPU implementation notes - since this is small and all of the accesses are constant,
        //I think I can create a per thread array and the compiler will put them on registers
        //which will be nice and efficient - however, will need to use memory profiler to make
        //sure this is actually happening, becasue if it isn't I want to switch to a shared memory
        //scheme.
        
        energy += coeff * fcut_all * Tn_ij[ powers[0] ] * Tn_ik[ powers[1] ] * Tn_jk[ powers[2] ];    

        deriv[0] = fcut[0] * Tnd_ij[ powers[0] ] + fcutderiv[0] * Tn_ij[ powers[0] ];
        deriv[1] = fcut[1] * Tnd_ik[ powers[1] ] + fcutderiv[1] * Tn_ik[ powers[1] ];
        deriv[2] = fcut[2] * Tnd_jk[ powers[2] ] + fcutderiv[2] * Tn_jk[ powers[2] ];

        // deriv will need to be created on the GPU in the same way as powers, which ever way ends up being more efficient.


        force_scalar[0]  = coeff * deriv[0] * fcut_2[0] * Tn_ik[powers[1]]  * Tn_jk[powers[2]] ;
        force_scalar[1]  = coeff * deriv[1] * fcut_2[1] * Tn_ij[powers[0]]  * Tn_jk[powers[2]] ;
        force_scalar[2]  = coeff * deriv[2] * fcut_2[2] * Tn_ij[powers[0]]  * Tn_ik[powers[1]] ;

        // force scalar will need to be created on the GPU in the same way as powers and deriv, which ever way ends up being more efficient.
        
        // Accumulate forces/stresses on/from the ij pair
        
        force[0*CHDIM+0] += force_scalar[0] * dr[0*CHDIM+0];
        force[0*CHDIM+1] += force_scalar[0] * dr[0*CHDIM+1];
        force[0*CHDIM+2] += force_scalar[0] * dr[0*CHDIM+2];

        force[1*CHDIM+0] -= force_scalar[0] * dr[0*CHDIM+0];
        force[1*CHDIM+1] -= force_scalar[0] * dr[0*CHDIM+1];
        force[1*CHDIM+2] -= force_scalar[0] * dr[0*CHDIM+2];   

        // dr2_3B looks like a function call, but the optimizer should remove it entirely.
#ifdef USE_DISTANCE_TENSOR
        // New stress code.
        stress[0] -= force_scalar[0]  * dr2_3B(dr2,0,0,0,0); // xx tensor component
        stress[1] -= force_scalar[0]  * dr2_3B(dr2,0,0,0,1); // xy tensor component
        stress[2] -= force_scalar[0]  * dr2_3B(dr2,0,0,0,2); // xz tensor component
        stress[3] -= force_scalar[0]  * dr2_3B(dr2,0,1,0,1); // yy tensor component
        stress[4] -= force_scalar[0]  * dr2_3B(dr2,0,1,0,2); // yz tensor component
        stress[5] -= force_scalar[0]  * dr2_3B(dr2,0,2,0,2); // zz tensor component
        
#else
        stress[0] -= force_scalar[0]  * dr[0*CHDIM+0] * dr[0*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[0]  * dr[0*CHDIM+0] * dr[0*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[0]  * dr[0*CHDIM+0] * dr[0*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[0]  * dr[0*CHDIM+1] * dr[0*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[0]  * dr[0*CHDIM+1] * dr[0*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[0]  * dr[0*CHDIM+2] * dr[0*CHDIM+2]; // zz tensor component
#endif        
        // Accumulate forces/stresses on/from the ik pair
        
        force[0*CHDIM+0] += force_scalar[1] * dr[1*CHDIM+0];
        force[0*CHDIM+1] += force_scalar[1] * dr[1*CHDIM+1];
        force[0*CHDIM+2] += force_scalar[1] * dr[1*CHDIM+2];

        force[2*CHDIM+0] -= force_scalar[1] * dr[1*CHDIM+0];
        force[2*CHDIM+1] -= force_scalar[1] * dr[1*CHDIM+1];
        force[2*CHDIM+2] -= force_scalar[1] * dr[1*CHDIM+2];   

#ifdef USE_DISTANCE_TENSOR
        stress[0] -= force_scalar[1]  * dr2_3B(dr2,1,0,1,0); // xx tensor component
        stress[1] -= force_scalar[1]  * dr2_3B(dr2,1,0,1,1); // xy tensor component
        stress[2] -= force_scalar[1]  * dr2_3B(dr2,1,0,1,2); // xz tensor component
        stress[3] -= force_scalar[1]  * dr2_3B(dr2,1,1,1,1); // yy tensor component
        stress[4] -= force_scalar[1]  * dr2_3B(dr2,1,1,1,2); // yz tensor component
        stress[5] -= force_scalar[1]  * dr2_3B(dr2,1,2,1,2); // zz tensor component
#else
        stress[0] -= force_scalar[1]  * dr[1*CHDIM+0] * dr[1*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[1]  * dr[1*CHDIM+0] * dr[1*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[1]  * dr[1*CHDIM+0] * dr[1*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[1]  * dr[1*CHDIM+1] * dr[1*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[1]  * dr[1*CHDIM+1] * dr[1*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[1]  * dr[1*CHDIM+2] * dr[1*CHDIM+2]; // zz tensor component
#endif
        
        // Accumulate forces/stresses on/from the jk pair
        
        force[1*CHDIM+0] += force_scalar[2] * dr[2*CHDIM+0];
        force[1*CHDIM+1] += force_scalar[2] * dr[2*CHDIM+1];
        force[1*CHDIM+2] += force_scalar[2] * dr[2*CHDIM+2];

        force[2*CHDIM+0] -= force_scalar[2] * dr[2*CHDIM+0];
        force[2*CHDIM+1] -= force_scalar[2] * dr[2*CHDIM+1];
        force[2*CHDIM+2] -= force_scalar[2] * dr[2*CHDIM+2];   

#ifdef USE_DISTANCE_TENSOR
        stress[0] -= force_scalar[2]  * dr2_3B(dr2,2,0,2,0); // xx tensor component
        stress[1] -= force_scalar[2]  * dr2_3B(dr2,2,0,2,1); // xy tensor component
        stress[2] -= force_scalar[2]  * dr2_3B(dr2,2,0,2,2); // xz tensor component
        stress[3] -= force_scalar[2]  * dr2_3B(dr2,2,1,2,1); // yy tensor component
        stress[4] -= force_scalar[2]  * dr2_3B(dr2,2,1,2,2); // yz tensor component
        stress[5] -= force_scalar[2]  * dr2_3B(dr2,2,2,2,2); // zz tensor component
#else        
        stress[0] -= force_scalar[2]  * dr[2*CHDIM+0] * dr[2*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[2]  * dr[2*CHDIM+0] * dr[2*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[2]  * dr[2*CHDIM+0] * dr[2*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[2]  * dr[2*CHDIM+1] * dr[2*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[2]  * dr[2*CHDIM+1] * dr[2*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[2]  * dr[2*CHDIM+2] * dr[2*CHDIM+2]; // zz tensor component
#endif        
    } */

    //force_scalar_in[0] = force_scalar[0];
    //force_scalar_in[1] = force_scalar[1];
    //force_scalar_in[2] = force_scalar[2];

    // the last iteration needs to be removed from
    // the gpu so force scalars can be returned.
    // unclear from meeting with professor if this 
    // actually works correctly or if anyone is using this,
    // so may end up getting removed.

    force_scalar_in[0] = 1;
    force_scalar_in[1] = 1;
    force_scalar_in[2] = 1;

    // what actually is the physical meaning behind returning the last force scalars to force scalar in?

    return;    
}

void chimesFF::compute_4B(const vector<double> & dx, const vector<double> & dr, const vector<int> & typ_idxs, vector<double> & force, vector<double> & stress, double & energy, chimes4BTmp &tmp)
{              
        vector<double> dummy_force_scalar(6);
        compute_4B(dx, dr, typ_idxs, force, stress, energy, tmp, dummy_force_scalar);                                                               
}
void chimesFF::compute_4B(const vector<double> & dx, const vector<double> & dr, const vector<int> & typ_idxs, vector<double> & force, vector<double> & stress, double & energy, chimes4BTmp &tmp, vector<double> & force_scalar_in)
{
    // Compute 4b (input: 4 atoms or distances, corresponding types... outputs (updates) force, acceleration, energy, stress
    //
    // Input parameters:
    //
    // dx_ij: Scalar (pair distance)
    // dr_ij: 1d-Array (pair distance: [x, y, and z-component])
    // Force: [natoms in interaction set][x,y, and z-component] *note
    // Stress [sxx, sxy, sxz, syy, syz, szz]
    // Energy: Scalar; energy for interaction set
    // Tmp: Structure containing temporary data.
    // Assumes atom indices start from zero
    // Assumes distances are atom_2 - atom_1
    //
    // *note: force and dr are packed vectors of coordinates.

    const int natoms = 4;                     // Number of atoms in an interaction set
    const int npairs = natoms*(natoms-1)/2;    // Number of pairs in an interaction set


    double fcut[npairs] ;
    double fcutderiv[npairs] ;
    //double deriv[npairs] ;
    

#if DEBUG == 1  
    if ( force.size() != CHDIM * natoms ) {
        cout << "Error: force vector had incorrect dimension of " << force.size() << endl ;
        exit(1) ;
    }
#endif      

    vector<double> &Tn_ij   = tmp.Tn_ij ;
    vector<double> &Tn_ik   = tmp.Tn_ik ;
    vector<double> &Tn_il   = tmp.Tn_il ;
    vector<double> &Tn_jk   = tmp.Tn_jk ;
    vector<double> &Tn_jl   = tmp.Tn_jl ;
    vector<double> &Tn_kl   = tmp.Tn_kl ;        
                                          
    vector<double> &Tnd_ij  = tmp.Tnd_ij ;
    vector<double> &Tnd_ik  = tmp.Tnd_ik ;
    vector<double> &Tnd_il  = tmp.Tnd_il ;  
    vector<double> &Tnd_jk  = tmp.Tnd_jk ;
    vector<double> &Tnd_jl  = tmp.Tnd_jl ;
    vector<double> &Tnd_kl  = tmp.Tnd_kl ;              

    int idx = typ_idxs[0]*natmtyps*natmtyps*natmtyps
        + typ_idxs[1]*natmtyps*natmtyps + typ_idxs[2]*natmtyps + typ_idxs[3] ;

    int quadidx = atom_int_quad_map[idx] ;

    if(quadidx < 0)    // Skipping an excluded interaction
        return;

    vector<int> & mapped_pair_idx = pair_int_quad_map[idx] ;

    // Check whether cutoffs are within allowed ranges

    for(int i=0; i<npairs; i++)
        if (dx[i] >= chimes_4b_cutoff[ quadidx ][1][mapped_pair_idx[i]])
            return;    

    // At this point, all distances are within allowed ranges. We can now proceed to the force/stress/energy calculation
    
    // Set up the polynomials
    
    set_cheby_polys(Tn_ij, Tnd_ij, dx[0], atom_int_pair_map[ typ_idxs[0]*natmtyps + typ_idxs[1] ], chimes_4b_cutoff[quadidx][0][mapped_pair_idx[0]], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[0]], 2);
    set_cheby_polys(Tn_ik, Tnd_ik, dx[1], atom_int_pair_map[ typ_idxs[0]*natmtyps + typ_idxs[2] ], chimes_4b_cutoff[quadidx][0][mapped_pair_idx[1]], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[1]], 2);
    set_cheby_polys(Tn_il, Tnd_il, dx[2], atom_int_pair_map[ typ_idxs[0]*natmtyps + typ_idxs[3] ], chimes_4b_cutoff[quadidx][0][mapped_pair_idx[2]], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[2]], 2);
    set_cheby_polys(Tn_jk, Tnd_jk, dx[3], atom_int_pair_map[ typ_idxs[1]*natmtyps + typ_idxs[2] ], chimes_4b_cutoff[quadidx][0][mapped_pair_idx[3]], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[3]], 2);
    set_cheby_polys(Tn_jl, Tnd_jl, dx[4], atom_int_pair_map[ typ_idxs[1]*natmtyps + typ_idxs[3] ], chimes_4b_cutoff[quadidx][0][mapped_pair_idx[4]], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[4]], 2);
    set_cheby_polys(Tn_kl, Tnd_kl, dx[5], atom_int_pair_map[ typ_idxs[2]*natmtyps + typ_idxs[3] ], chimes_4b_cutoff[quadidx][0][mapped_pair_idx[5]], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[5]], 2);     
    
#ifdef USE_DISTANCE_TENSOR  
    // Tensor product of displacement vectors.
    double dr2[CHDIM*CHDIM*npairs*npairs] ;
    init_distance_tensor(dr2, dr, npairs) ;
#endif
    
    
    // Set up the smoothing functions
    for (int i=0; i<npairs; i++)    
        get_fcut(dx[i], chimes_4b_cutoff[quadidx][1][mapped_pair_idx[i]], fcut[i], fcutderiv[i]);


    // Product of all 6 fcuts.
    double fcut_all = fcut[0] * fcut[1] * fcut[2] * fcut[3] * fcut[4] * fcut[5]  ;

    // Product of 5 fcuts divided by dx.
    double fcut_5[npairs] ;
    fcut_5[0] = fcut[1] * fcut[2] * fcut[3] * fcut[4] * fcut[5] / dx[0] ;
    fcut_5[1] = fcut[0] * fcut[2] * fcut[3] * fcut[4] * fcut[5] / dx[1] ;
    fcut_5[2] = fcut[0] * fcut[1] * fcut[3] * fcut[4] * fcut[5] / dx[2] ;
    fcut_5[3] = fcut[0] * fcut[1] * fcut[2] * fcut[4] * fcut[5] / dx[3] ;
    fcut_5[4] = fcut[0] * fcut[1] * fcut[2] * fcut[3] * fcut[5] / dx[4] ;
    fcut_5[5] = fcut[0] * fcut[1] * fcut[2] * fcut[3] * fcut[4] / dx[5] ;
    
    // Start the force/stress/energy calculation

    // Begin setting up the pointers for the GPU.

    double *device_chimes_params, *device_force, *device_stress;
    int *device_chimes_pows;

    struct poly_pointers_4b device_polys;

    // Various malloc calls
    hipMalloc(&device_chimes_params, chimes_4b_params[quadidx].size() * sizeof(double));
    hipMalloc(&device_force, force.size() * sizeof(double));
    hipMalloc(&device_stress, stress.size() * sizeof(double));
    hipMalloc(&device_chimes_pows, chimes_4b_powers_flat[quadidx].size() * sizeof(double));

    // Tn and Tnd mallocs
    hipMalloc(&device_polys.Tn_ij, Tn_ij.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_ik, Tn_ik.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_il, Tn_il.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_jk, Tn_jk.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_jl, Tn_jl.size() * sizeof(double));
    hipMalloc(&device_polys.Tn_kl, Tn_kl.size() * sizeof(double));

    hipMalloc(&device_polys.Tnd_ij, Tnd_ij.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_ik, Tnd_ik.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_il, Tnd_il.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_jk, Tnd_jk.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_jl, Tnd_jl.size() * sizeof(double));
    hipMalloc(&device_polys.Tnd_kl, Tnd_kl.size() * sizeof(double));

    // All of the mallocs should be done, can begin transferring memory
    // over to the gpu

    // Let's start off with constant memory first because
    // I just like it better for some reason.

    hipMemcpyToSymbol(HIP_SYMBOL(fcut_4b), fcut, npairs * sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(fcutderiv_4b), fcutderiv, npairs * sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(fcut5_4b), fcut_5, npairs * sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dr_4b), dr.data(), dr.size() * sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(pair_idx_4b), mapped_pair_idx.data(), mapped_pair_idx.size() * sizeof(int));
    #ifdef USE_DISTANCE_TENSOR
        hipMemcpyToSymbol(HIP_SYMBOL(dr2_4b), dr2, sizeof(dr2_4b));
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(gpu_energy), &energy, sizeof(double));

    hipMemcpy(device_chimes_params, chimes_4b_params[quadidx].data(), chimes_4b_params[quadidx].size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_force, force.data(), force.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_stress, stress.data(), stress.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_chimes_pows, chimes_4b_powers_flat[quadidx].data(), chimes_4b_powers_flat[quadidx].size() * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(device_polys.Tn_ij, Tn_ij.data(), Tn_ij.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_ik, Tn_ik.data(), Tn_ik.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_il, Tn_il.data(), Tn_il.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_jk, Tn_jk.data(), Tn_jk.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_jl, Tn_jl.data(), Tn_jl.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tn_kl, Tn_kl.data(), Tn_kl.size() * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(device_polys.Tnd_ij, Tnd_ij.data(), Tnd_ij.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_ik, Tnd_ik.data(), Tnd_ik.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_il, Tnd_il.data(), Tnd_il.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_jk, Tnd_jk.data(), Tnd_jk.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_jl, Tnd_jl.data(), Tnd_jl.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_polys.Tnd_kl, Tnd_kl.data(), Tnd_kl.size() * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 512;
    // Can start with this
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(ceil((double)ncoeffs_4b[quadidx]/blockSize), 1 , 1);

    // Start the kernel
    compute4b_helper<<<dimGrid, dimBlock>>>(ncoeffs_4b[quadidx], fcut_all, device_chimes_params, device_chimes_pows, device_force, device_stress, device_polys);
    hipDeviceSynchronize();

    // retrieve relevant results from the GPU

    hipMemcpyFromSymbol(&energy, HIP_SYMBOL(gpu_energy), sizeof(double));
    hipMemcpy(stress.data(), device_stress, stress.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(force.data(), device_force, force.size() * sizeof(double), hipMemcpyDeviceToHost);

    // Clean up time.

    hipFree(device_chimes_params);
    hipFree(device_chimes_pows);
    hipFree(device_force);
    hipFree(device_stress);

    hipFree(device_polys.Tn_ij);
    hipFree(device_polys.Tn_ik);
    hipFree(device_polys.Tn_il);
    hipFree(device_polys.Tn_jk);
    hipFree(device_polys.Tn_jl);
    hipFree(device_polys.Tn_kl);

    hipFree(device_polys.Tnd_ij);
    hipFree(device_polys.Tnd_ik);
    hipFree(device_polys.Tnd_il);
    hipFree(device_polys.Tnd_jk);
    hipFree(device_polys.Tnd_jl);
    hipFree(device_polys.Tnd_kl);

    // all clean I think.

    /*
    for(int coeffs=0; coeffs<ncoeffs_4b[quadidx]; coeffs++)
    {
        coeff = chimes_4b_params[quadidx][coeffs];
        
        for (int i=0; i<npairs; i++)
            powers[i] = chimes_4b_powers[quadidx][coeffs][mapped_pair_idx[i]];

        double Tn_ij_ik_il =  Tn_ij[ powers[0] ] * Tn_ik[ powers[1] ] * Tn_il[ powers[2] ] ;
        double Tn_jk_jl    =  Tn_jk[ powers[3] ] * Tn_jl[ powers[4] ] ;
        double Tn_kl_5     =  Tn_kl[ powers[5] ] ;

        energy += coeff * fcut_all * Tn_ij_ik_il * Tn_jk_jl * Tn_kl_5 ;        

        deriv[0] = fcut[0] * Tnd_ij[ powers[0] ] + fcutderiv[0] * Tn_ij[ powers[0] ];
        deriv[1] = fcut[1] * Tnd_ik[ powers[1] ] + fcutderiv[1] * Tn_ik[ powers[1] ];
        deriv[2] = fcut[2] * Tnd_il[ powers[2] ] + fcutderiv[2] * Tn_il[ powers[2] ];
        deriv[3] = fcut[3] * Tnd_jk[ powers[3] ] + fcutderiv[3] * Tn_jk[ powers[3] ];
        deriv[4] = fcut[4] * Tnd_jl[ powers[4] ] + fcutderiv[4] * Tn_jl[ powers[4] ];
        deriv[5] = fcut[5] * Tnd_kl[ powers[5] ] + fcutderiv[5] * Tn_kl[ powers[5] ];        

        force_scalar[0]  = coeff * deriv[0] * fcut_5[0] * Tn_ik[powers[1]]  * Tn_il[powers[2]] * Tn_jk_jl * Tn_kl_5 ;
        force_scalar[1]  = coeff * deriv[1] * fcut_5[1] * Tn_ij[powers[0]]  * Tn_il[powers[2]] * Tn_jk_jl * Tn_kl_5 ;
        force_scalar[2]  = coeff * deriv[2] * fcut_5[2] * Tn_ij[powers[0]]  * Tn_ik[powers[1]] * Tn_jk_jl * Tn_kl_5 ;
        force_scalar[3]  = coeff * deriv[3] * fcut_5[3] * Tn_ij_ik_il  * Tn_jl[powers[4]] * Tn_kl_5 ;
        force_scalar[4]  = coeff * deriv[4] * fcut_5[4] * Tn_ij_ik_il  * Tn_jk[powers[3]] * Tn_kl_5 ;
        force_scalar[5]  = coeff * deriv[5] * fcut_5[5] * Tn_ij_ik_il * Tn_jk_jl ;

        // Accumulate forces/stresses on/from the ij pair
        
        force[0*CHDIM+0] += force_scalar[0] * dr[0*CHDIM+0];
        force[0*CHDIM+1] += force_scalar[0] * dr[0*CHDIM+1];
        force[0*CHDIM+2] += force_scalar[0] * dr[0*CHDIM+2];

        force[1*CHDIM+0] -= force_scalar[0] * dr[0*CHDIM+0];
        force[1*CHDIM+1] -= force_scalar[0] * dr[0*CHDIM+1];
        force[1*CHDIM+2] -= force_scalar[0] * dr[0*CHDIM+2];   

#ifdef USE_DISTANCE_TENSOR      
        stress[0] -= force_scalar[0]  * dr2_4B(dr2,0,0,0,0); // xx tensor component
        stress[1] -= force_scalar[0]  * dr2_4B(dr2,0,0,0,1); // xy tensor component
        stress[2] -= force_scalar[0]  * dr2_4B(dr2,0,0,0,2); // xz tensor component
        stress[3] -= force_scalar[0]  * dr2_4B(dr2,0,1,0,1); // yy tensor component
        stress[4] -= force_scalar[0]  * dr2_4B(dr2,0,1,0,2); // yz tensor component
        stress[5] -= force_scalar[0]  * dr2_4B(dr2,0,2,0,2); // zz tensor component
#else
        stress[0] -= force_scalar[0]  * dr[0*CHDIM+0] * dr[0*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[0]  * dr[0*CHDIM+0] * dr[0*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[0]  * dr[0*CHDIM+0] * dr[0*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[0]  * dr[0*CHDIM+1] * dr[0*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[0]  * dr[0*CHDIM+1] * dr[0*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[0]  * dr[0*CHDIM+2] * dr[0*CHDIM+2]; // zz tensor component
#endif      
        
        // Accumulate forces/stresses on/from the ik pair
        
        force[0*CHDIM+0] += force_scalar[1] * dr[1*CHDIM+0];
        force[0*CHDIM+1] += force_scalar[1] * dr[1*CHDIM+1];
        force[0*CHDIM+2] += force_scalar[1] * dr[1*CHDIM+2];

        force[2*CHDIM+0] -= force_scalar[1] * dr[1*CHDIM+0];
        force[2*CHDIM+1] -= force_scalar[1] * dr[1*CHDIM+1];
        force[2*CHDIM+2] -= force_scalar[1] * dr[1*CHDIM+2];   

#if USE_DISTANCE_TENSOR     
        stress[0] -= force_scalar[1]  * dr2_4B(dr2,1,0,1,0); // xx tensor component
        stress[1] -= force_scalar[1]  * dr2_4B(dr2,1,0,1,1); // xy tensor component
        stress[2] -= force_scalar[1]  * dr2_4B(dr2,1,0,1,2); // xz tensor component
        stress[3] -= force_scalar[1]  * dr2_4B(dr2,1,1,1,1); // yy tensor component
        stress[4] -= force_scalar[1]  * dr2_4B(dr2,1,1,1,2); // yz tensor component
        stress[5] -= force_scalar[1]  * dr2_4B(dr2,1,2,1,2); // zz tensor component
#else        
        stress[0] -= force_scalar[1]  * dr[1*CHDIM+0] * dr[1*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[1]  * dr[1*CHDIM+0] * dr[1*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[1]  * dr[1*CHDIM+0] * dr[1*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[1]  * dr[1*CHDIM+1] * dr[1*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[1]  * dr[1*CHDIM+1] * dr[1*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[1]  * dr[1*CHDIM+2] * dr[1*CHDIM+2]; // zz tensor component
#endif      
        // Accumulate forces/stresses on/from the il pair
        
        force[0*CHDIM+0] += force_scalar[2] * dr[2*CHDIM+0];
        force[0*CHDIM+1] += force_scalar[2] * dr[2*CHDIM+1];
        force[0*CHDIM+2] += force_scalar[2] * dr[2*CHDIM+2];

        force[3*CHDIM+0] -= force_scalar[2] * dr[2*CHDIM+0];
        force[3*CHDIM+1] -= force_scalar[2] * dr[2*CHDIM+1];
        force[3*CHDIM+2] -= force_scalar[2] * dr[2*CHDIM+2];   

#ifdef USE_DISTANCE_TENSOR        
        stress[0] -= force_scalar[2]  * dr2_4B(dr2,2,0,2,0); // xx tensor component
        stress[1] -= force_scalar[2]  * dr2_4B(dr2,2,0,2,1); // xy tensor component
        stress[2] -= force_scalar[2]  * dr2_4B(dr2,2,0,2,2); // xz tensor component
        stress[3] -= force_scalar[2]  * dr2_4B(dr2,2,1,2,1); // yy tensor component
        stress[4] -= force_scalar[2]  * dr2_4B(dr2,2,1,2,2); // yz tensor component
        stress[5] -= force_scalar[2]  * dr2_4B(dr2,2,2,2,2); // zz tensor component           
#else       
        stress[0] -= force_scalar[2]  * dr[2*CHDIM+0] * dr[2*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[2]  * dr[2*CHDIM+0] * dr[2*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[2]  * dr[2*CHDIM+0] * dr[2*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[2]  * dr[2*CHDIM+1] * dr[2*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[2]  * dr[2*CHDIM+1] * dr[2*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[2]  * dr[2*CHDIM+2] * dr[2*CHDIM+2]; // zz tensor component           
#endif
        
        // Accumulate forces/stresses on/from the jk pair
        
        force[1*CHDIM+0] += force_scalar[3] * dr[3*CHDIM+0];
        force[1*CHDIM+1] += force_scalar[3] * dr[3*CHDIM+1];
        force[1*CHDIM+2] += force_scalar[3] * dr[3*CHDIM+2];

        force[2*CHDIM+0] -= force_scalar[3] * dr[3*CHDIM+0];
        force[2*CHDIM+1] -= force_scalar[3] * dr[3*CHDIM+1];
        force[2*CHDIM+2] -= force_scalar[3] * dr[3*CHDIM+2];   

#ifdef USE_DISTANCE_TENSOR      
        stress[0] -= force_scalar[3]  * dr2_4B(dr2,3,0,3,0); // xx tensor component
        stress[1] -= force_scalar[3]  * dr2_4B(dr2,3,0,3,1); // xy tensor component
        stress[2] -= force_scalar[3]  * dr2_4B(dr2,3,0,3,2); // xz tensor component
        stress[3] -= force_scalar[3]  * dr2_4B(dr2,3,1,3,1); // yy tensor component
        stress[4] -= force_scalar[3]  * dr2_4B(dr2,3,1,3,2); // yz tensor component
        stress[5] -= force_scalar[3]  * dr2_4B(dr2,3,2,3,2); // zz tensor component
#else
        stress[0] -= force_scalar[3]  * dr[3*CHDIM+0] * dr[3*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[3]  * dr[3*CHDIM+0] * dr[3*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[3]  * dr[3*CHDIM+0] * dr[3*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[3]  * dr[3*CHDIM+1] * dr[3*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[3]  * dr[3*CHDIM+1] * dr[3*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[3]  * dr[3*CHDIM+2] * dr[3*CHDIM+2]; // zz tensor component
#endif
        
        // Accumulate forces/stresses on/from the jl pair
        
        force[1*CHDIM+0] += force_scalar[4] * dr[4*CHDIM+0];
        force[1*CHDIM+1] += force_scalar[4] * dr[4*CHDIM+1];
        force[1*CHDIM+2] += force_scalar[4] * dr[4*CHDIM+2];

        force[3*CHDIM+0] -= force_scalar[4] * dr[4*CHDIM+0];
        force[3*CHDIM+1] -= force_scalar[4] * dr[4*CHDIM+1];
        force[3*CHDIM+2] -= force_scalar[4] * dr[4*CHDIM+2];     

#ifdef USE_DISTANCE_TENSOR      
        stress[0] -= force_scalar[4]  * dr2_4B(dr2,4,0,4,0); // xx tensor component
        stress[1] -= force_scalar[4]  * dr2_4B(dr2,4,0,4,1); // xy tensor component
        stress[2] -= force_scalar[4]  * dr2_4B(dr2,4,0,4,2); // xz tensor component
        stress[3] -= force_scalar[4]  * dr2_4B(dr2,4,1,4,1); // yy tensor component
        stress[4] -= force_scalar[4]  * dr2_4B(dr2,4,1,4,2); // yz tensor component
        stress[5] -= force_scalar[4]  * dr2_4B(dr2,4,2,4,2); // zz tensor component
#else       
        stress[0] -= force_scalar[4]  * dr[4*CHDIM+0] * dr[4*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[4]  * dr[4*CHDIM+0] * dr[4*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[4]  * dr[4*CHDIM+0] * dr[4*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[4]  * dr[4*CHDIM+1] * dr[4*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[4]  * dr[4*CHDIM+1] * dr[4*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[4]  * dr[4*CHDIM+2] * dr[4*CHDIM+2]; // zz tensor component
#endif      
        // Accumulate forces/stresses on/from the kl pair
        
        force[2*CHDIM+0] += force_scalar[5] * dr[5*CHDIM+0];
        force[2*CHDIM+1] += force_scalar[5] * dr[5*CHDIM+1];
        force[2*CHDIM+2] += force_scalar[5] * dr[5*CHDIM+2];

        force[3*CHDIM+0] -= force_scalar[5] * dr[5*CHDIM+0];
        force[3*CHDIM+1] -= force_scalar[5] * dr[5*CHDIM+1];
        force[3*CHDIM+2] -= force_scalar[5] * dr[5*CHDIM+2];     

#ifdef USE_DISTANCE_TENSOR
        stress[0] -= force_scalar[5]  * dr2_4B(dr2,5,0,5,0); // xx tensor component
        stress[1] -= force_scalar[5]  * dr2_4B(dr2,5,0,5,1); // xy tensor component
        stress[2] -= force_scalar[5]  * dr2_4B(dr2,5,0,5,2); // xz tensor component
        stress[3] -= force_scalar[5]  * dr2_4B(dr2,5,1,5,1); // yy tensor component
        stress[4] -= force_scalar[5]  * dr2_4B(dr2,5,1,5,2); // yz tensor component
        stress[5] -= force_scalar[5]  * dr2_4B(dr2,5,2,5,2); // zz tensor component
#else       
        stress[0] -= force_scalar[5]  * dr[5*CHDIM+0] * dr[5*CHDIM+0]; // xx tensor component
        stress[1] -= force_scalar[5]  * dr[5*CHDIM+0] * dr[5*CHDIM+1]; // xy tensor component
        stress[2] -= force_scalar[5]  * dr[5*CHDIM+0] * dr[5*CHDIM+2]; // xz tensor component
        stress[3] -= force_scalar[5]  * dr[5*CHDIM+1] * dr[5*CHDIM+1]; // yy tensor component
        stress[4] -= force_scalar[5]  * dr[5*CHDIM+1] * dr[5*CHDIM+2]; // yz tensor component
        stress[5] -= force_scalar[5]  * dr[5*CHDIM+2] * dr[5*CHDIM+2]; // zz tensor component
#endif      
    }
    
	force_scalar_in[0] = force_scalar[0];
	force_scalar_in[1] = force_scalar[1];
	force_scalar_in[2] = force_scalar[2];
	force_scalar_in[3] = force_scalar[3];
	force_scalar_in[4] = force_scalar[4];
	force_scalar_in[5] = force_scalar[5];
    */

    force_scalar_in[0] = 1;
	force_scalar_in[1] = 1;
	force_scalar_in[2] = 1;
	force_scalar_in[3] = 1;
	force_scalar_in[4] = 1;
	force_scalar_in[5] = 1;
   
    return;
}

void chimesFF::get_cutoff_2B(vector<vector<double> >  & cutoff_2b)
{
    int dim = chimes_2b_cutoff.size();
    
    cutoff_2b.resize(dim);
    
    for (int i=0; i<dim; i++)
    {
        cutoff_2b[i].resize(0);
        
        for (int j=0; j<chimes_2b_cutoff[i].size(); j++)
        
            cutoff_2b[i].push_back(chimes_2b_cutoff[i][j]);
    }
}

double chimesFF::max_cutoff(int ntypes, vector<vector<vector<double> > > & cutoff_list)
{
    double max = cutoff_list[0][1][0]; 
    
    for (int i=0; i<ntypes; i++)
        for (int j=0; j<cutoff_list[i][1].size(); j++)
            if (cutoff_list[i][1][j] > max)
                max = cutoff_list[i][1][j];

    return max;

}

double chimesFF::max_cutoff_2B(bool silent)
{
    double max = chimes_2b_cutoff[0][1]; 
    
    for (int i=0; i<chimes_2b_cutoff.size(); i++)
        if (chimes_2b_cutoff[i][1] > max)
            max = chimes_2b_cutoff[i][1];
    
    if ((rank == 0)&&(!silent))        
        cout << "chimesFF: " << "\t" << "Setting 2-body max cutoff to: " << max << endl;
    
    return max;    
}

double chimesFF::max_cutoff_3B(bool silent)
{
    
    if (poly_orders[1] == 0)
        return 0.0;
    
    double max = max_cutoff(chimes_3b_cutoff.size(), chimes_3b_cutoff);
    
    if ((rank == 0)&&(!silent))    
        cout << "chimesFF: " << "\t" << "Setting 3-body max cutoff to: " << max << endl;
    
    return max;
    
}

double chimesFF::max_cutoff_4B(bool silent)
{
    if (poly_orders[2] == 0)
        return 0.0;
    
    double max =  max_cutoff(chimes_4b_cutoff.size(), chimes_4b_cutoff);
        
    if ((rank == 0)&&(!silent))    
        cout << "chimesFF: " << "\t" << "Setting 4-body max cutoff to: " << max << endl;
    
    return max;
}

void chimesFF::set_atomtypes(vector<string> & type_list)
{
    type_list.resize(natmtyps);
    
    for(int i=0;i<natmtyps;i++)
        type_list[i] = atmtyps[i];
}

int chimesFF::get_atom_pair_index(int pair_id)
{
    return atom_idx_pair_map[pair_id];
}

void chimesFF::build_pair_int_quad_map()
{
    // Build the pair maps for all possible quads.  Moved build_atom_and_pair_mappers out of the compute_XX routines
    // to support GPU environment without string operations.
    // This must be called prior to force evaluation.

    const int natoms = 4 ;
    const int npairs = natoms * (natoms-1) / 2 ;
    vector<int> pair_map(npairs) ;
    vector<int> typ_idxs(natoms) ;

    if ( atom_int_quad_map.size() == 0 ) return ; // No quads !
    
    pair_int_quad_map.resize(natmtyps*natmtyps*natmtyps*natmtyps) ;

    
    for ( int i = 0 ; i < natmtyps ; i++ )
    {
        typ_idxs[0] = i ;
        for ( int j = 0 ; j < natmtyps ; j++ )
        {
            typ_idxs[1] = j ;
            for ( int k = 0 ; k < natmtyps ; k++ )
            {
                typ_idxs[2] = k ;
                for ( int l = 0 ; l < natmtyps ; l++ )
                {
                    typ_idxs[3] = l ;
                    int idx = i*natmtyps*natmtyps*natmtyps + j*natmtyps*natmtyps + k*natmtyps + l ;
                    int quadidx = atom_int_quad_map[idx];

                    // Skip excluded interactions
                    if (quadidx < 0)
                        continue;

                    build_atom_and_pair_mappers(natoms, npairs, typ_idxs, quad_params_pair_typs[quadidx], pair_map);

                    // Save for re-use in force evaluators.
                    if ( quadidx >= natmtyps * natmtyps * natmtyps * natmtyps )
                    {
                        cout << "Error: quadidx out of range\n" ;
                        cout << "Quadidx = " << quadidx << endl ;
                        exit(1) ;
                    }

                    // Note: The entire vector<> is copied and stored.                  
                    pair_int_quad_map[idx] = pair_map ;
                }
            }
        }
    }
    for ( int i = 0 ; i < pair_int_quad_map.size() ; i++ )
    {
        if ( pair_int_quad_map[i].size() == 0 )
        {
		if (atom_int_quad_map[i] >= 0)
            		cout << "Error: Did not initialize pair_int_quad_map for entry " << i << endl ;
		else
			cout << "Warning: Did not initialize pair_int_quad_map for excluded entry " << i << endl ;
        }
    }   
}

void chimesFF::build_pair_int_trip_map()
// Build the pair maps for all possible triplets.  Moved build_atom_and_pair_mappers out of the compute_XX routines
// to support GPU environment without string operations.
// This must be called prior to force evaluation.
{
    const int natoms = 3 ;
    const int npairs = natoms * (natoms-1) / 2 ;
    vector<int> pair_map(npairs) ;
    vector<int> typ_idxs(natoms) ;

    if ( atom_int_trip_map.size() == 0 ) return ; // No trips !
    
    pair_int_trip_map.resize(natmtyps*natmtyps*natmtyps) ;
    
    for ( int i = 0 ; i < natmtyps ; i++ )
    {
        typ_idxs[0] = i ;
        for ( int j = 0 ; j < natmtyps ; j++ )
        {
            typ_idxs[1] = j ;
            for ( int k = 0 ; k < natmtyps ; k++ )
            {
                typ_idxs[2] = k ;
                int tripidx = atom_int_trip_map[i*natmtyps*natmtyps + j*natmtyps + k];
		
		// Skip excluded interactions
		if (tripidx < 0)
			continue;

                build_atom_and_pair_mappers(natoms, npairs, typ_idxs, trip_params_pair_typs[tripidx], pair_map);
                    
                // Save for re-use in force evaluators.
                if ( tripidx >= natmtyps * natmtyps * natmtyps * natmtyps )
                {
                    cout << "Error: tripidx out of range\n" ;
                    cout << "Tripidx = " << tripidx << endl ;
                    exit(1) ;
                }

                // Note: The entire vector<> is copied and stored.
                pair_int_trip_map[i*natmtyps*natmtyps + j*natmtyps + k] = pair_map ;
            }
        }
    }
    for ( int i = 0 ; i < pair_int_trip_map.size() ; i++ )
    {
        if ( pair_int_trip_map[i].size() == 0 )
        {
		if (atom_int_trip_map[i] >= 0)
            		cout << "Error: Did not initialize pair_int_trip_map for entry " << i << endl ;
		else
			cout << "Warning: Did not initialize pair_int_trip_map for excluded entry " << i << endl ;
        }
    }
    
}

